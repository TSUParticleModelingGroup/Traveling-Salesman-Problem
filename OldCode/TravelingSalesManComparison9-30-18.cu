#include "hip/hip_runtime.h"
//Maybe add a random pertibation
//I don't like that the wall strength is just an arbitrary large number
//work on pointer in exact
//Move the point off (0,0) before running the extrution program
//Move wall by a set number because the wall will not be a fixed distance apart
//
// Set optimal radius off of average seperation
// Clean up extrution
// If min distance is 1 find max distance then how big would a grid be that fit this. Use this to find optimal radius
//
//nvcc TravelingSalesManComparison9-30-18.cu -o TSPCompare093018 -lglut -lGL -lm
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#define PI 3.14159265359
#define BIG_NUMBER 1000000.0

#define BLOCK 256

#define X_WINDOW 800
#define Y_WINDOW 800

#define X_MAX (1.1)
#define X_MIN (-1.1)

#define Y_MAX (1.1)
#define Y_MIN (-1.1)

#define RANDOM_FILL_SHAPE_CIRCLE -1
#define RANDOM_FILL_SHAPE_SQUARE 1

#define TIME_STEP_SIZE 0.002
#define STEPS_BETWEEN_VIEWING 100

#define NUMBER_OF_WALL_MOVES 1000
#define RADIUS_STEP_SIZE 0.001
#define TIME_BETWEEN_WALL_MOVES 1.0

#define RELAX_TIME 10.0

#define WALL_STRENGTH 20000.0

#define SLOPE_REPULSION -100.0
#define	SLOPE_ATRACTION -1.0

#define DAMP 20.0 
#define MASS 80.0

#define FLOAT_ROUND_OFF 0.000001

#define DRAW_EXHAUSTIVE_PATH 1
#define DRAW_NEAREST_NEIGHBOR_PATH -1
#define DRAW_NBODY_EXTRUSION_PATH 1
#define DELAY_TO_RECORD -1
#define PRINT_EDGE_COST -1
#define PRINT_EXHAUSTIVE_PATHS -1
#define PRINT_PATHS -1
#define PRINT_RAW_DATA_FILE 1

// Globals
FILE *RawDataFile;
FILE *StatsFile;

//Function prototypes
float x_machine_to_x_screen(int x);
float y_machine_to_y_screen(int y);
float x_machine_to_x_world(int x);
float y_machine_to_y_world(int y);
float x_world_to_x_screen(float x);
float y_world_to_y_screen(float y);
void openRawDataFile(int scope, int numberOfRuns);
void placeNodesRandom(float4 *pos, unsigned int srandSeed, int scope, int n);
int checkNodes(float4 *pos, int n);
void placeNodesGrid(float4 *pos, int rows, int columns);
void getNumberOfNodesFromNodeFile(int *numberOfNodes);
float adjustingNodes(float4 *pos, int numberOfNodes);
void placeNodesFromAFile(float4 *pos, int *numberOfNodes);
void printEdgeCosts(float4 *pos, int n, float nodeAdjustmentFactor);
int factorial(int n);
void printPathOrder(int* path, int n);
float getPathCost(int *path, float4 *pos, int type, int n);
void swap(int *path, int i, int j);
void heappermute(int* path, int m, float4 *pos, int* exhaustivePath, float* minCost, int n);
float exhaustiveTSP(float4 *pos, int* exhaustivePath, int n);
float nearestNeighborTSP(float4 *pos, int* path, int n);
void setNbodyInitailConditions(float4 *nodePos, float4 *pos, float4 *vel, float* mass, int n);
float4 adjustSoGeometricCenterIsZero(float4 *pos, int n);
float4 FindGeometericCenter(float4 *pos, int n);
float findAverageSeperation(float4 *pos, int n);
float findOptimalOuterRadius(float4 *pos, int n);
void drawPoints(float4 pos, int n);
void drawNbodyExtrusion(float4 *pos, float4 center, float outerRadius, float innerRadius, float optimalOuterRadius, int inOut, int n);
void getPathNbody(float4 *pos, float4 geometericCenter, int* path, int n);
float NbodyExtrusionTSP(float4 *nodePos, float4 *pos, float4 *vel, float4 *acc, float* mass, int* path, int n);
void drawFInalPicture(float4 *pos, int *pathA, int *pathB, int *pathC, int scope, int n);
void getInputFromUser(int* scope, int* numberOfNodes, int* numberOfRuns, int* maxNumberOfRows, int* maxNumberOfColumns, unsigned int* srandSeed);
void control();

float x_machine_to_x_screen(int x)
{
	return( (2.0*x)/X_WINDOW-1.0 );
}

float y_machine_to_y_screen(int y)
{
	return( -(2.0*y)/Y_WINDOW+1.0 );
}

/*	Takes machine x and y which start in the upper left corner and go from zero to X_WINDOW
	left to right and form zero to Y_WINDOW top to bottom and transslates this into world
	points which are a X_MIN to X_MAX, Y_MIN to Y_MAX window.
*/
float x_machine_to_x_world(int x)
{
	float range;
	range = X_MAX - X_MIN;
	return( (range/X_WINDOW)*x + X_MIN );
}

float y_machine_to_y_world(int y)
{
	float range;
	range = Y_MAX - Y_MIN;
	return(-((range/Y_WINDOW)*y - Y_MAX));
}

/*	Take world  points to screen points
*/
float x_world_to_x_screen(float x)
{
	float range;
	range = X_MAX - X_MIN;
	return( -1.0 + 2.0*(x - X_MIN)/range );
}

float y_world_to_y_screen(float y)
{
	float range;
	range = Y_MAX - Y_MIN;
	return( -1.0 + 2.0*(y - Y_MIN)/range );
}

void openRawDataFile(int scope, int numberOfRuns)
{
	char tagName[50];
	char fileName[256];
	
	strcpy(tagName,"");
	if(scope == 2)
	{
		strcat(tagName,"Random_TSP_Raw_Data_CSV");
	}
	if(scope == 4)
	{
		strcat(tagName,"Grid_TSP_Raw_Data_CSV");
	}
	
	snprintf(fileName, 256, "%s", tagName);
	
	RawDataFile = fopen(fileName, "wb");
  	
	fprintf(RawDataFile, "  Number of runs %d\n",numberOfRuns);
	
	if(scope == 2)
	{
		fprintf(RawDataFile, "  Run  Nodes  Exact  NNeighbor  NBody\n\n");
	}
	if(scope == 4)
	{
		fprintf(RawDataFile, "  Run  Rows  Columns  Exact  NNeighbor  NBody\n\n");
	}
}

void placeNodesRandom(float4 *pos, unsigned int srandSeed, int scope, int n)
{
	time_t t;
	float xStart, xStop, yStart, yStop, radius, maxRadius;
	float mag, seperation;
	int repeatedNodeTest;
	
	xStart = -1.0;
	xStop  = 1.0;
	yStart = -1.0;
	yStop  = 1.0;
	
	maxRadius = 1.0;

	srand((unsigned) time(&t));

	if(scope == 1) srand(srandSeed);
	else srand((unsigned) time(&t));

	if(RANDOM_FILL_SHAPE_SQUARE == 1)
	{
		for(int i = 0; i < n; i++)
		{
			repeatedNodeTest = 0;
			while(repeatedNodeTest == 0)
			{
				pos[i].x = (float)rand()/((float)RAND_MAX)*(xStop - xStart) + xStart;
				pos[i].y = (float)rand()/((float)RAND_MAX)*(yStop - yStart) + yStart;
				repeatedNodeTest = 1;
				for(int j = 0; j < i; j++)
				{
					seperation = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
					if(seperation < FLOAT_ROUND_OFF)
					{
						repeatedNodeTest = 0;
						break;
					}
				}
			}
		}
	}
	
	if(RANDOM_FILL_SHAPE_CIRCLE == 1)
	{
		for(int i = 0; i < n; i++)
		{
			repeatedNodeTest = 0;
			while(repeatedNodeTest == 0)
			{
				pos[i].x = (float)rand()/((float)RAND_MAX)*(xStop - xStart) + xStart;
				pos[i].y = (float)rand()/((float)RAND_MAX)*(yStop - yStart) + yStart;
				mag = sqrt(pos[i].x*pos[i].x + pos[i].y*pos[i].y);
				radius = ((float)rand()/(float)RAND_MAX)*maxRadius;
				pos[i].x *= radius/mag;
				pos[i].y *= radius/mag;
				repeatedNodeTest = 1;
				for(int j = 0; j < i; j++)
				{
					seperation = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
					if(seperation < FLOAT_ROUND_OFF)
					{
						repeatedNodeTest = 0;
						break;
					}
				}
			}
		}
	}
}

int checkNodes(float4 *pos, int n)
{
	float seperation;
	
	for(int i = 0; i < (n - 1); i++)
	{
		for(int j = (i + 1); j < n; j++)
		{
			seperation = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
			if(seperation < FLOAT_ROUND_OFF)
			{
				return(-1);
			}
		}
	}
	return(1);
}

void placeNodesGrid(float4 *pos, int rows, int columns)
{
	int i,j,k;
	float dx, dy;
	float xStart, yStart;
	
	xStart = 0.0;
	yStart = 0.0;
	dx = 1.0;
	dy = 1.0;
	
	k = 0;
	for(i = 0; i < columns; i++)
	{
		for(j = 0; j < rows; j++)
		{
			pos[k].x = xStart + dx*i;
			pos[k].y = yStart + dy*j;
			k++;
		}
	}
}

void getNumberOfNodesFromNodeFile(int *numberOfNodes)
{
	FILE *nodeFile;
	nodeFile = fopen("nodeFile", "rb");
	fscanf(nodeFile,"%d", numberOfNodes);
	fclose(nodeFile);
}

void placeNodesFromAFile(float4 *pos, int *numberOfNodes)
{
	FILE *nodeFile;
	int nodeNumber;
	nodeFile = fopen("nodeFile", "rb");
	fscanf(nodeFile,"%d", numberOfNodes);
	
	for(int i = 0; i < *numberOfNodes; i++)
	{
		fscanf(nodeFile,"%d %f %f", &nodeNumber, &pos[i].x, &pos[i].y );
		//printf("\n\nnode number= %d, %f, %f\n\n", nodeNumber, pos[i].x, pos[i].y);
	}
	
	fclose(nodeFile);
}

//This function adjustes the nodes so that the minimal seperation is 1.0
float adjustingNodes(float4 *pos, int numberOfNodes)
{
	float temp;
	float nodeAdjustmentFactor;
	float minSeperation;
	float dx,dy;
	int i,j;
	
	dx = pos[1].x-pos[2].x;
	dy = pos[1].y-pos[2].y;
	minSeperation = sqrt(dx*dx + dy*dy);
	for(i = 0; i < numberOfNodes; i++)
	{
		for(j = i + 1; j < numberOfNodes; j++)
		{
			dx = pos[i].x-pos[j].x;
			dy = pos[i].y-pos[j].y;
			temp = sqrt(dx*dx + dy*dy);
			if(temp < minSeperation)
			{
				minSeperation = temp;
			}
		}
	}
	
	nodeAdjustmentFactor = 1.0/minSeperation;
	
	for(int i = 0; i < numberOfNodes; i++)
	{
		pos[i].x = pos[i].x*nodeAdjustmentFactor;
		pos[i].y = pos[i].y*nodeAdjustmentFactor;
	}
	
	return(nodeAdjustmentFactor);
}

float findDistanceToOuterMostNode(float4 *pos, int numberOfNodes)
{
	float temp;
	float distanceToOutermostNode = 0.0;
	
	for(int i = 0; i < numberOfNodes; i++)
	{
		temp = sqrt(pos[i].x*pos[i].x + pos[i].y*pos[i].y);
		if(temp > distanceToOutermostNode)
		{
			distanceToOutermostNode = temp;
		}
	}
	return(distanceToOutermostNode);
}

void printEdgeCosts(float4 *pos, int n, float nodeAdjustmentFactor)
{
	float temp;
	for(int i = 0; i < n; i++)
	{
		for(int j = i + 1; j < n; j++)
		{	
			temp = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y))/nodeAdjustmentFactor;
			printf("edge cost [%d, %d] = %f\n", i, j, temp);
		}
	}
}

int factorial(int n)
{
	int outPut = n;
	
	for(int i = n-1; i > 0; i--)
	{
		outPut *= i;	
	}
	return(outPut);
}

void printPathOrder(int* path, int n)
{
	printf("  ");
	for(int i = 0; i < n-1; i++)
	{
		printf("%d->", path[i]);	
	}
	printf("%d", path[n-1]);
}

float getPathCost(int *path, float4 *pos, int type, int n)
{
	float cost;
	int i, j, k;
	
	//Checking path validaty 
	for(i = 0; i < n; i++)
	{
		if(path[i] < 0 || (n-1) < path[i])
		{
			printf("\n\n  Error -> Path out of range! Type = %d", type);
			printf("\n  path[%d] = %d\n\n", i, path[i]);
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		
		for(j = 0; j < i; j++)
		{
			if(path[i] == path[j])
			{
				printf("\n\n Error -> Path has a repeated index! Type = %d\n", type);
				printPathOrder(path, n);
				printf("\n\n");
				printf("\n\n  Good Bye.  \n\n");
				exit(0);
			}
		}
	}
	
	cost = 0.0;
	for(k = 0; k < n-1; k++)
	{
		i = path[k];
		j = path[k+1];
		cost += sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
	}
	i = path[n-1];
	j = path[0];
	cost += sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
	
	return(cost);
}

void swap(int *path, int i, int j)
{
	int temp;
	temp = path[i];
	path[i] = path[j];
	path[j] = temp;
}

void heappermute(int *path, int m, float4 *pos, int *exhaustivePath, float *minCost, int n) 
{
	int i;
	float pathCost;
	int* pathPlus = (int*)malloc(n*sizeof(int));

	if (m == 1) 
	{
		pathPlus[0] = 0;
		for(i = 1; i < n; i++)
		{
			pathPlus[i] = path[i-1];	
		}
		
		pathCost = getPathCost(pathPlus, pos, 1, n);
		
		if(PRINT_EXHAUSTIVE_PATHS == 1)
		{
			printf("\n");
			printPathOrder(pathPlus, n);
			printf(" cost = %f", pathCost);
		}
		
		if(pathCost < minCost[0])
		{
			minCost[0] = pathCost;
			for(i = 0; i < n; i++)
			{
				exhaustivePath[i] = pathPlus[i];	
			}
		}
    	}
	else 
	{
		for (i = 0; i < m; i++) 
		{
			heappermute(path, m-1, pos, exhaustivePath, minCost, n);
			if (m % 2 == 1) 
			{
				swap(path, 0, m-1);
			}
			else 
			{
				swap(path, i, m-1);
			}
		}
	}
	free(pathPlus);
}

float exhaustiveTSP(float4 *pos, int* exhaustivePath, int n)
{
	float cost[1];
	int* path = (int*)malloc((n-1)*sizeof(int));
	
	exhaustivePath[0] = 0;
	for(int i = 1; i < n; i++)
	{
		exhaustivePath[i] = i;
		path[i-1] = i;	
	}
	cost[0] = getPathCost(exhaustivePath, pos, 1, n);
	
	heappermute(path, n-1, pos, exhaustivePath, cost, n);
	free(path);
	return(cost[0]);
}

float nearestNeighborTSP(float4 *pos, int* path, int n)
{
	int i, j, k, nextNode, nodeFound;
	float minCost, pathCost, edgeCost, maxEdgeCost;
	int* used = (int*)malloc(n*sizeof(int));
	
	maxEdgeCost = 0.0;
	for(i = 0; i < n; i++)
	{
		for(j = 0; j < n; j++)
		{
			edgeCost = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
			if(edgeCost > maxEdgeCost) 
			{
				maxEdgeCost = edgeCost;
			}	
		}	
	}
	maxEdgeCost += 1.0;
	
	for(i = 0; i < n; i++)
	{
		used[i] = -1;	
	}
	
	path[0] = 0;
	used[0] = 1;
	
	k = 0;
	
	minCost = maxEdgeCost;
	while(k < n-1)
	{
		nodeFound = 0;
		for(j = 0; j < n; j++)
		{
			i = path[k];
			edgeCost = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
			if(edgeCost <= minCost && used[j] == -1)
			{
				minCost = edgeCost;
				nextNode = j;
				nodeFound = 1;
			}	
		}
		if(nodeFound == 0)
		{
			printf("\n\n  There was a problem in the nearest neighbor function. No next node was found.\n\n");
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		nodeFound = 0;
		
		k++;
		path[k] = nextNode;
		used[nextNode] = 1;
		minCost = maxEdgeCost;
	}
	
	pathCost = getPathCost(path, pos, 2, n);
	free(used);
	return(pathCost);
}

void setNbodyInitailConditions(float4 *nodePos, float4 *pos, float4 *vel, float* mass, int n)
{
	int i;

	for(i = 0; i < n; i++)
	{
		pos[i].x = nodePos[i].x;
		pos[i].y = nodePos[i].y;
		
		vel[i].x = 0.0;
		vel[i].y = 0.0;
		
		mass[i] = MASS;
	}
}

void moveAnyNodeOffDeadCenter(float4 *pos, int n)
{
	int i;

	for(i = 0; i < n; i++)
	{
		if( sqrt(pos[i].x*pos[i].x + pos[i].y*pos[i].y) < 0.001) 
		{
			pos[i].x = 0.001;
			pos[i].y = 0.001;
		}
	}
}

float4 adjustSoGeometricCenterIsZero(float4 *pos, int n)
{
	float4 geometricCenter;
	
	geometricCenter.x = 0.0;
	geometricCenter.y = 0.0;
	geometricCenter.z = 0.0;
	geometricCenter.w = 0.0;
	
	for(int i = 0; i < n; i++)
	{
		geometricCenter.x += pos[i].x;
		geometricCenter.y += pos[i].y;
	}
	
	geometricCenter.x /= (float)n;
	geometricCenter.y /= (float)n;
	
	for(int i = 0; i < n; i++)
	{
		pos[i].x -= geometricCenter.x;
		pos[i].y -= geometricCenter.y;
	}
	return(geometricCenter);
}

float4 FindGeometericCenter(float4 *pos, int n)
{
	float4 massCenter;
	
	massCenter.x = 0.0;
	massCenter.y = 0.0;
	massCenter.z = 0.0;
	massCenter.w = 0.0;
	
	for(int i = 0; i < n; i++)
	{
		massCenter.x += pos[i].x*MASS;
		massCenter.y += pos[i].y*MASS;
	}
	
	massCenter.x /= (float)n*MASS;
	massCenter.y /= (float)n*MASS;
	
	return(massCenter);
}

float findOptimalOuterRadius(float4 *pos, int n)
{
	float maxRadius;
	float perimeter;
	int *nearestNeighborPath;
	
	//The min node seperation is 1. So this lets all node fit on the circle with min seperation.
	//maxRadius = (n*1.0)/(2.0*PI);
	
	// Setting outer radius to be a fraction of the nearest nieghbor length
	nearestNeighborPath = (int*)malloc((n)*sizeof(int));
	perimeter = nearestNeighborTSP(pos, nearestNeighborPath, n);
	maxRadius = 0.9*perimeter/(2.0*PI);
	free(nearestNeighborPath);
	
	
	return(maxRadius);
}

float findAverageSeperation(float4 *pos, int n)
{
	float averageSeperation;
	float totalSeperation = 0.0;
	
	for(int i = 0; i < n; i++)
	{
		for(int j = i + 1; j < n; j++)
		{
			totalSeperation += sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
		}
	}
	averageSeperation = ((float)(n*n - n))/2.0;
	
	return(averageSeperation);
}

void drawPoints(float4 *pos, int n)
{
	int i;
	float outerRadius = findDistanceToOuterMostNode(pos, n);
	float normalizingFactor = outerRadius; //((float)n)/IDEAL_NUMBER_OF_NODES;
	
	glClear(GL_COLOR_BUFFER_BIT);
	
	glPointSize(5.0);
	glColor3f(1.0,0.0,0.0);
	for(i = 0; i < n; i++)
	{
		glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(pos[i].x/normalizingFactor),y_world_to_y_screen(pos[i].y/normalizingFactor));
		glEnd();

	}
	glFlush();
}

void drawNbodyExtrusion(float4 *pos, float4 center, float outerRadius, float innerRadius, float optimalOuterRadius, int inOut, int n)
{
	int i;
	int lineAmount = 100;
	float normalizingFactor = outerRadius;
	
	optimalOuterRadius/= normalizingFactor;
	outerRadius /= normalizingFactor;
	innerRadius /= normalizingFactor;

	glClear(GL_COLOR_BUFFER_BIT);
	
	GLfloat twicePi = 2.0f * PI;
	
	glLineWidth(1.0);
	glColor3f(1.0,1.0,1.0);
	glBegin(GL_LINE_LOOP);
		for(i = 0; i <= lineAmount;i++) 
		{ 
			glVertex2f(x_world_to_x_screen(center.x + (optimalOuterRadius*cos(i*twicePi/lineAmount))), 
			           y_world_to_y_screen(center.y + (optimalOuterRadius*sin(i*twicePi/lineAmount))));
		}
	glEnd();
	
	glLineWidth(1.0);
	if(inOut == -1) glColor3f(1.0,0.0,0.0);
	else glColor3f(0.0,1.0,0.0);
	glBegin(GL_LINE_LOOP);
		for(i = 0; i <= lineAmount;i++) 
		{ 
			glVertex2f(x_world_to_x_screen(center.x + (outerRadius*cos(i*twicePi/lineAmount))), 
			           y_world_to_y_screen(center.y + (outerRadius*sin(i*twicePi/lineAmount))));
		}
	glEnd();
	
	glLineWidth(1.0);
	glColor3f(1.0,1.0,0.0);
	glBegin(GL_LINE_LOOP);
		for(i = 0; i <= lineAmount;i++) 
		{ 
			glVertex2f(x_world_to_x_screen(center.x + (innerRadius*cos(i*twicePi/lineAmount))), 
			           y_world_to_y_screen(center.y + (innerRadius*sin(i*twicePi/lineAmount))));
		}
	glEnd();
	
	glPointSize(5.0);
	glColor3f(1.0,0.0,0.0);
	for(i = 0; i < n; i++)
	{
		glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(pos[i].x/normalizingFactor),y_world_to_y_screen(pos[i].y/normalizingFactor));
		glEnd();

	}
	
	glFlush();
}

__global__ void accelerationsNbody(float4 *nodePos, float4 *pos, float4 *vel, float4 *acc, float *mass, float4 geometericCenter, float innerRadius, float outerRadius, int n)
{
	int j,ii;
    //float3 bodyBodyForce, wallForce, forceSum;
    float3 forceSum;
    float4 nodePosMe, posMe;
    float dx, dy, d, edgeLength; 
    //float dc, temp;
    float radius, forceMag;
    __shared__ float4 shNodePos[BLOCK], shPos[BLOCK];
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    
    forceSum.x = 0.0;
	forceSum.y = 0.0;
	
	nodePosMe.x = nodePos[id].x;
	nodePosMe.y = nodePos[id].y;
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
		    
    for(j=0; j < gridDim.x; j++)
    {
    	if(threadIdx.x + blockDim.x*j < n)
    	{
    		shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
    		shNodePos[threadIdx.x] = nodePos[threadIdx.x + blockDim.x*j];
    	}
    	__syncthreads();
   
		#pragma unroll 32
        for(int i = 0; i < blockDim.x; i++)	
        {
        	ii = i + blockDim.x*j;
		    if(ii != id && ii < n) 
		    {
				dx = shPos[i].x - posMe.x;
				dy = shPos[i].y - posMe.y;
				d = sqrtf(dx*dx + dy*dy);
				//dc = (sqrtf(posMe.x*posMe.x + posMe.y*posMe.y) + sqrtf(shPos[i].x*shPos[i].x + shPos[i].y*shPos[i].y))/2.0;
				//temp = (outerRadius - dc)/outerRadius;
				//dc = abs(shPos[i].x*posMe.y - shPos[i].y*posMe.x)/d;
				edgeLength = sqrtf((shNodePos[i].x - nodePosMe.x)*(shNodePos[i].x - nodePosMe.x) + (shNodePos[i].y - nodePosMe.y)*(shNodePos[i].y - nodePosMe.y));
				
				//if(dc < 0.5*innerRadius) forceMag = 0.0;
				if(d <= edgeLength)
				{
					forceMag = (edgeLength - d)*SLOPE_REPULSION;
	
				}
				else if(d <= 1.2*edgeLength)
				{
					//forceMag = 25.0*temp*(edgeLength - d)*SLOPE_ATRACTION;
					forceMag = (edgeLength - d)*SLOPE_ATRACTION;
					//forceMag = 0.0;
				}
				else
				{
					forceMag = 0.0;
				}
				
				forceSum.x += forceMag*dx/d;
				forceSum.y += forceMag*dy/d;
		    }
		}
	}
	
	if(id < n)
	{
		// Forces between node and the walls
		dx = posMe.x - geometericCenter.x;
		dy = posMe.y - geometericCenter.y; 
		radius = sqrtf(dx*dx + dy*dy);
	
		if(radius < innerRadius) // Inside inner wall
		{
			forceMag = WALL_STRENGTH*(innerRadius - radius);
			forceSum.x += forceMag*dx/radius;
			forceSum.y += forceMag*dy/radius;
		}
		else if(radius > outerRadius) // Outside outer wall
		{
			forceMag = WALL_STRENGTH*(outerRadius - radius);
			forceSum.x += forceMag*dx/radius;
			forceSum.y += forceMag*dy/radius;
		}
		
		// Adding on damping force.
		forceSum.x += -DAMP*vel[id].x;
		forceSum.y += -DAMP*vel[id].y;
		
		// Creating the accelerations.
	    acc[id].x = forceSum.x/mass[id];
	    acc[id].y = forceSum.y/mass[id];
    }
}

__global__ void moveNbody(float4 *pos, float4 *vel, float4 *acc, float dt, int n)
{
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    if(id < n)
    {
	    vel[id].x += acc[id].x*dt;
		vel[id].y += acc[id].y*dt;
		
		pos[id].x  += vel[id].x*dt;
		pos[id].y  += vel[id].y*dt;
    }
}

void getPathNbody(float4 *pos, float4 geometericCenter, int* path, int n)
{
	int i;
	float minValue;
	float *angle = (float*)malloc(n*sizeof(float));
	int *used = (int*)malloc(n*sizeof(int));
	
	for(i = 0; i < n; i++)
	{
		if(pos[i].x == 0 && pos[i].y == 0)
		{
			angle[i] = 0.0;
		}
		else if(pos[i].x >= 0 && pos[i].y >= 0)
		{
			if(pos[i].x == 0) angle[i] = 90.0;
			else angle[i] = atan(pos[i].y/pos[i].x)*180.0/PI;
		}
		else if(pos[i].x < 0 && pos[i].y >= 0)
		{
			angle[i] = 180.0 - atan(pos[i].y/(-pos[i].x))*180.0/PI;
		}
		else if(pos[i].x <= 0 && pos[i].y < 0)
		{
			if(pos[i].x == 0) angle[i] = 270.0;
			else angle[i] = 180.0 + atan(pos[i].y/pos[i].x)*180.0/PI;
		}
		else
		{
			angle[i] = 360.0 - atan(-pos[i].y/pos[i].x)*180.0/PI;
		}
	}
	
	for(i = 0; i < n; i++)
	{
		used[i] = 0;
	}
	
	for(int k = 0; k < n; k++)
	{
		minValue = 400.0;
		for(i = 0; i < n; i++)
		{
			if(angle[i] < minValue && used[i] == 0)
			{
				minValue = angle[i];
				path[k] = i;
			}
		}
		used[path[k]] = 1;
		//printf("path[%d] = %d\n", k, path[k]);
	}
	
	free(angle);
	free(used);
}

float NbodyExtrusionTSP(float4 *nodePos, float4 *pos, float4 *vel, float4 *acc, float* mass, int* path, int n)
{
	int draw_count;
	int outerWallDirection;
	float dr;
	float dt = TIME_STEP_SIZE;
	float pathCost;
	float time;
	float4 geometericCenter;
	float innerRadius, outerRadius, optimalOuterRadius;
	float averageSeperation;
	
	dim3 block, grid;
	float4 *posGPU, *velGPU, *accGPU; 
	float *massGPU;
	float4 *nodePosGPU;
	int wallMovesLeft;
	float nodeDensity, optimalDensity;
	
	// Setting up GPU parrellel structure.
	block.x = BLOCK;
	block.y = 1;
	block.z = 1;
	
	grid.x = (n-1)/block.x + 1;
	grid.y = 1;
	grid.z = 1;
	
	// Allocating memory.
	hipMalloc( (void**)&nodePosGPU, n *sizeof(float4));
	hipMalloc( (void**)&posGPU, n *sizeof(float4));
	hipMalloc( (void**)&velGPU, n *sizeof(float4));
	hipMalloc( (void**)&accGPU, n *sizeof(float4));
	hipMalloc( (void**)&massGPU, n *sizeof(float));
	
	// This is used to pause the program so you can setup to take a video of a run.
	if(DELAY_TO_RECORD == 1)
	{
		printf("\n\n  Enter a character to start\n\n"); getchar();
	}
	
	// Copying information up to the GPU.
	hipMemcpy( nodePosGPU, nodePos, n *sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( posGPU, pos, n *sizeof(float4), hipMemcpyHostToDevice );
    hipMemcpy( velGPU, vel, n *sizeof(float4), hipMemcpyHostToDevice );
    hipMemcpy( massGPU, mass, n *sizeof(float), hipMemcpyHostToDevice );
    
    // Moving the nodes into a circle.
    averageSeperation = findAverageSeperation(nodePos, n);
    optimalOuterRadius = findOptimalOuterRadius(nodePos, n);
	dr = (optimalOuterRadius - 0.0)/(float)NUMBER_OF_WALL_MOVES;
    outerWallDirection = -1;
    innerRadius = 0.0;
    outerRadius = findDistanceToOuterMostNode(pos, n);
    geometericCenter = FindGeometericCenter(pos, n);
	drawNbodyExtrusion(pos, geometericCenter, outerRadius, innerRadius, optimalOuterRadius, outerWallDirection, n);
	//optimalDensity = 2.23; //based off of 4X4 grid
	//nodeDensity = ((float)n)/(PI*outerRadius*outerRadius);
	time = 0.0;
	draw_count = 0;
	while(optimalOuterRadius/2.0 < outerRadius)
	{
		outerRadius -= dr;
		while(time < TIME_BETWEEN_WALL_MOVES)
		{		
			accelerationsNbody<<<grid, block>>>(nodePosGPU, posGPU, velGPU, accGPU, massGPU, geometericCenter, innerRadius, outerRadius, n);
			moveNbody<<<grid, block>>>(posGPU, velGPU, accGPU, dt, n);
			
			if(draw_count == STEPS_BETWEEN_VIEWING)
			{
				hipMemcpy( pos, posGPU, n *sizeof(float4), hipMemcpyDeviceToHost );
				drawNbodyExtrusion(pos, geometericCenter, outerRadius, innerRadius, optimalOuterRadius, outerWallDirection, n);
				draw_count = 0;
			}
			draw_count++;
			time += dt;
		}
		
		//nodeDensity = ((float)n)/(PI*outerRadius*outerRadius);
		time = 0.0;
		outerRadius -= dr;
	}
	
	wallMovesLeft = NUMBER_OF_WALL_MOVES;
	outerWallDirection = -1;
	while(innerRadius + 0.01 < optimalOuterRadius)
	{
		while(time < TIME_BETWEEN_WALL_MOVES)
		{		
			accelerationsNbody<<<grid, block>>>(nodePosGPU, posGPU, velGPU, accGPU, massGPU, geometericCenter, innerRadius, outerRadius, n);
			moveNbody<<<grid, block>>>(posGPU, velGPU, accGPU, dt, n);
			
			if(draw_count == STEPS_BETWEEN_VIEWING)
			{
				hipMemcpy( pos, posGPU, n *sizeof(float4), hipMemcpyDeviceToHost );
				drawNbodyExtrusion(pos, geometericCenter, outerRadius, innerRadius, optimalOuterRadius, outerWallDirection, n);
				draw_count = 0;
			}
			draw_count++;
			time += dt;
		}
		
		// Reseting run conditions.
		time = 0.0;
		innerRadius += dr;
		//wallMovesLeft--;
		//printf("\n Number of moves left = %d", wallMovesLeft);
		if((outerRadius - innerRadius) < 1.0) //averageSeperation/2.0);
		{ 
			outerWallDirection = 1.0;
		}
		if(outerRadius < optimalOuterRadius) outerRadius += dr*outerWallDirection;
	}
	
	getPathNbody(pos, geometericCenter, path, n);
	pathCost = getPathCost(path, nodePos, 3, n);
	
	return(pathCost);
}

void drawFInalPicture(float4 *pos, int *pathA, int *pathB, int *pathC, int scope, int n)
{	
	int i;
	float outerRadius = findDistanceToOuterMostNode(pos, n);
	float normalizingFactor = outerRadius; //((float)n)/IDEAL_NUMBER_OF_NODES;

	glClear(GL_COLOR_BUFFER_BIT);
	
	//exhuastivePath path
	if(scope == 1 || scope == 2)
	{
		if(DRAW_EXHAUSTIVE_PATH == 1)
		{
			glLineWidth(4.0);
			glColor3f(0.0,0.0,1.0);
			glBegin(GL_LINE_LOOP);
				for(i = 0; i < n; i++)
				{
					glVertex2f(x_world_to_x_screen(pos[pathA[i]].x/normalizingFactor),y_world_to_y_screen(pos[pathA[i]].y/normalizingFactor));
				}
			glEnd();
		}
	}
	
	//Nearest Neighbor path
	if(DRAW_NEAREST_NEIGHBOR_PATH == 1)
	{
		glLineWidth(6.0);
		glColor3f(0.0,1.0,0.0);
		glBegin(GL_LINE_LOOP);
			for(i = 0; i < n; i++)
			{
				glVertex2f(x_world_to_x_screen(pos[pathB[i]].x/normalizingFactor),y_world_to_y_screen(pos[pathB[i]].y/normalizingFactor));
			}
		glEnd();
	}
	
	//Nbody Extrusion path
	if(DRAW_NBODY_EXTRUSION_PATH == 1)
	{
		glLineWidth(4.0);
		glColor3f(1.0,0.0,0.0);
		glBegin(GL_LINE_LOOP);
			for(i = 0; i < n; i++)
			{
				glVertex2f(x_world_to_x_screen(pos[pathC[i]].x/normalizingFactor),y_world_to_y_screen(pos[pathC[i]].y/normalizingFactor));
			}
		glEnd();
	}
	
	//Placing nodes
	glPointSize(8.0);
	glColor3f(1.0,1.0,1.0);
	for(i = 0; i < n; i++)
	{
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(pos[i].x/normalizingFactor),y_world_to_y_screen(pos[i].y/normalizingFactor));
		glEnd();
	}
	
	//Nearest neighbor start node 
	if(DRAW_NEAREST_NEIGHBOR_PATH == 1)
	{
		glPointSize(10.0);
		glColor3f(0.0,0.0,1.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(pos[pathB[0]].x/normalizingFactor),y_world_to_y_screen(pos[pathB[0]].y/normalizingFactor));
		glEnd();
	}
	
	//Nbody extrution start and stop nodes
	if(DRAW_NBODY_EXTRUSION_PATH == 1)
	{
		glPointSize(10.0);
		glColor3f(0.0,1.0,0.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(pos[pathC[0]].x/normalizingFactor),y_world_to_y_screen(pos[pathC[0]].y/normalizingFactor));
		glEnd();
	
		glColor3f(1.0,0.0,0.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(pos[pathC[n-1]].x/normalizingFactor),y_world_to_y_screen(pos[pathC[n-1]].y/normalizingFactor));
		glEnd();
	}
	
	glFlush();
}

void getInputFromUser(int* scope, int* numberOfNodes, int* numberOfRuns, int* maxNumberOfRows, int* maxNumberOfColumns, unsigned int* srandSeed)
{
	*scope = -1;
	*numberOfNodes = -1;
	*numberOfRuns = -1;
	*maxNumberOfRows = -1;
	*maxNumberOfColumns = -1;
	
	printf("\n\n  What type run would you like to perform?");
	printf("\n  1 for one small randomly generated run.");
	printf("\n  2 for a series of small randomly generated run.");
	printf("\n  3 for one on on a grid.");
	printf("\n  4 for a series of runs on randomly generated sized grids.");
	printf("\n  5 for a large randomly generated run.");
	printf("\n  6 to read nodes from nodeFile.");
	printf("\n\n  Inter an integer value: ");
	scanf("%d", scope);
	
	if(*scope == 1)
	{
		printf("\n\n  You will need to enter the number of nodes (Cities) wound you like to generate?");
		printf("\n  Note: If you choose a number bigger than 13 you may lock your computer up.");
		printf("\n        This is because we will be testing against the exaustive algorithim which has ((n-1)!)/2 paths.");
		
		printf("\n\n  You will also need to enter a seed to generate the random placed nodes");
		printf("\n  By intering the same seed on different exicutions of the program");
		printf("\n  you will be able to run multiple tests on the same node configuration.");
		
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of nodes) (seed)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", numberOfNodes, srandSeed);
		
		*numberOfRuns = 1;
	}
	else if(*scope == 2)
	{
		printf("\n\n  You will need to enter the number nodes (Cities) wound like to generate?");
		printf("\n  Note: If you choose a nuber bigger than 13 you may lock your computer up.");
		
		printf("\n\n  You will also need to enter how many randomly generated runs would you like to perform?");
		
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of nodes) (number of runs)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", numberOfNodes, numberOfRuns);
	}
	else if(*scope == 3)
	{
		printf("\n  You will need to enter the number of rows and columns for the grid you will generate.");
	
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of rows) (number of columns)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", maxNumberOfRows, maxNumberOfColumns);
		
		*numberOfRuns = 1;
	}
	else if(*scope == 4)
	{
		printf("\n  You will need to enter the number of rows and columns for the grid you will generate");
		printf("\n  and the number of runs");
	
		printf("\n  Enter three positive integer values seperated by a space: \n  (max number of rows) (max number of columns) (number of runs)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d %d", maxNumberOfRows, maxNumberOfColumns, numberOfRuns);
	}
	else if(*scope == 5)
	{
		printf("\n\n  You will need to enter the number of nodes (Cities) wound you like to generate?");
		printf("\n  Enter a positive integer values: \n  (number of nodes)");
		printf("\n\n  Inter your values: ");
		scanf("%d", numberOfNodes);
		
		*numberOfRuns = 1;
	}
	else if(*scope == 6)
	{	
		*numberOfRuns = 1;
	}
	else
	{
		printf("\n\n  Your input for type of run was invalide.");
		printf("\n\n  Good Bye.  \n\n");
		exit(0);
	}
}

void control()
{
	time_t t;
	int scope, numberOfNodes, numberOfRuns, maxNumberOfRows, maxNumberOfColumns;
	unsigned int srandSeed;
	int rows, columns, done;
	float4 *pos;
	float nodeAdjustmentFactor;
	float4 geometricCenter;
	float distanceToOutermostNode;
	int *exhaustivePath, *nearestNeighborPath, *NbodyExtrusionPath;
	float4 *posNbody, *velNbody, *accNbody; 
	float *massNbody;
	float exhaustiveCost, nearestNeighborCost, NbodyExtrusionCost;
	int nodeCheck;
	float temp;
	
	getInputFromUser(&scope, &numberOfNodes, &numberOfRuns, &maxNumberOfRows, &maxNumberOfColumns, &srandSeed);
	
	if(scope == 2 || scope == 4 && PRINT_RAW_DATA_FILE == 1)
	{
		openRawDataFile(scope, numberOfRuns);
	}
	
	float totalNearestNeighborCost = 0.0;
	float totalNbodyExtrusionCost = 0.0;
	float totalPercentErrorNearestNeighbor = 0.0;
	float totalPercentErrorNbodyExtrusion = 0.0;
	float NbodyExtrusionVSNearestNeighbor = 0.0;
	
	for(int i = 0; i < numberOfRuns; i++)
	{	
		printf("\n\n\n  ********************* Intermediate Run %d ********************* ", i+1);
		
		exhaustiveCost = BIG_NUMBER;
		nearestNeighborCost = BIG_NUMBER;
		NbodyExtrusionCost = BIG_NUMBER;
		
		//Alocating memory
		if(scope == 3)
		{
			rows = maxNumberOfRows;
			columns = maxNumberOfColumns;
			numberOfNodes = rows*columns;
		}	
		if(scope == 4)
		{
			srand((unsigned) time(&t));
			done = -1;
			while(done == -1)
			{
				rows = 1 + (float)rand()/((float)RAND_MAX)*maxNumberOfRows;
				columns = 1 + (float)rand()/((float)RAND_MAX)*maxNumberOfColumns;
				printf("\n\n  rows = %d columns = %d", rows, columns);
				numberOfNodes = rows*columns;
				if(rows == 1 && columns == 1) done = -1;
				else done = 1;
			}
		}
		if(scope == 6)
		{
			getNumberOfNodesFromNodeFile(&numberOfNodes);
		}
		
		pos = (float4*)malloc((numberOfNodes)*sizeof(float4));
	
		exhaustivePath = (int*)malloc((numberOfNodes)*sizeof(int));
		nearestNeighborPath = (int*)malloc((numberOfNodes)*sizeof(int));
		NbodyExtrusionPath = (int*)malloc((numberOfNodes)*sizeof(int));

		posNbody = (float4*)malloc((numberOfNodes)*sizeof(float4));
		velNbody = (float4*)malloc((numberOfNodes)*sizeof(float4));
		accNbody = (float4*)malloc((numberOfNodes)*sizeof(float4));
		massNbody = (float*)malloc((numberOfNodes)*sizeof(float4));
		
		//Creating nodes
		if(scope == 1 || scope == 2 || scope == 5)
		{	
			placeNodesRandom(pos, srandSeed, scope, numberOfNodes);
		}
		else if(scope == 3 || scope == 4)
		{
			placeNodesGrid(pos, rows, columns);
		}
		else if(scope == 6)
		{
			placeNodesFromAFile(pos, &numberOfNodes);
		}
		
		//Adjusting nodes
		geometricCenter = adjustSoGeometricCenterIsZero(pos, numberOfNodes);
		printf("\n\n  The geometric center of the nodes = (%f, %f)", geometricCenter.x, geometricCenter.y);
		
		distanceToOutermostNode = findDistanceToOuterMostNode(pos, numberOfNodes);
		printf("\n  The distance to the outermost node from the geometric center pre adjustment is %f", distanceToOutermostNode);
		
		nodeAdjustmentFactor = adjustingNodes(pos, numberOfNodes);
		printf("\n  The node adjustment factor = %f", nodeAdjustmentFactor);
		
		distanceToOutermostNode = findDistanceToOuterMostNode(pos, numberOfNodes);
		printf("\n  The distance to the outermost node from the geometric center post adjustment is %f", distanceToOutermostNode);
		
		//Checking to see if a node is repeated
		nodeCheck = checkNodes(pos, numberOfNodes);
		if(nodeCheck == -1)
		{
			printf("\n\n  There is a repeated node. Check your data set.");
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		
		//Drawing the adjusted nodes on the screen.
		drawPoints(pos, numberOfNodes); 
		
		//Printing the edge costs (lengths in this case)
		if(PRINT_EDGE_COST == 1)
		{
			printEdgeCosts(pos, numberOfNodes, nodeAdjustmentFactor);
		}
		
		//Finding exact cost
		printf("\n\n  Determining the exact cost.");
		if(scope == 1 || scope == 2)
		{	
			exhaustiveCost = exhaustiveTSP(pos, exhaustivePath, numberOfNodes);
		}
		else if(scope == 3 || scope == 4)
		{
			//Assuming all edges are the same length. So just get the length of the first edge.
			temp = sqrt((pos[0].x-pos[1].x)*(pos[0].x-pos[1].x) + (pos[0].y-pos[1].y)*(pos[0].y-pos[1].y));
			if(rows == 1 || columns == 1)
			{
				exhaustiveCost = temp*2.0*(numberOfNodes - 1);
			}
			else if(rows%2 == 0 || columns%2 == 0)
			{
				exhaustiveCost = temp*numberOfNodes;
			}
			else
			{
				exhaustiveCost = temp*(numberOfNodes - 1.0 + sqrt(2.0));
			}
		}
		else if(scope == 5)
		{
			exhaustiveCost = -1.0;
		}
		else if(scope == 6)
		{
			if(numberOfNodes < 14)
			{
				exhaustiveCost = exhaustiveTSP(pos, exhaustivePath, numberOfNodes);
			}
			else
			{
				exhaustiveCost = -1.0;
			}
		}
		printf("\n  Determining the exact cost is done.");
		
		//Finding nearest neighbor cost
		printf("\n\n  Running the nearest nieghbor algorithm.");
		nearestNeighborCost = nearestNeighborTSP(pos, nearestNeighborPath, numberOfNodes);
		printf("\n  The nearest nieghbor algorithm is done.");
		
		//Running n-body extrusion code
		printf("\n\n  Running the N-body extrusion algorithm."); 
		printf("  \n"); //I had to enter this carage return so it would print the line above before it started the algorithm
		setNbodyInitailConditions(pos, posNbody, velNbody, massNbody, numberOfNodes);
		moveAnyNodeOffDeadCenter(posNbody, numberOfNodes);
		NbodyExtrusionCost = NbodyExtrusionTSP(pos, posNbody, velNbody, accNbody, massNbody, NbodyExtrusionPath, numberOfNodes);
		printf("  The N-body extrusion algorithm is done.");
		
		//Unadjusting costs
		exhaustiveCost /= nodeAdjustmentFactor;
		nearestNeighborCost /= nodeAdjustmentFactor;
		NbodyExtrusionCost /= nodeAdjustmentFactor;
		
		totalNearestNeighborCost += nearestNeighborCost;
		totalNbodyExtrusionCost += NbodyExtrusionCost;
		
		//Sanity check
		if(nearestNeighborCost < exhaustiveCost - FLOAT_ROUND_OFF)
		{
			printf("\n\n  Nearest Neighbor cost (%f) is smaller than exhaustive cost (%f). Something is wrong!\n",nearestNeighborCost, exhaustiveCost);
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		if(NbodyExtrusionCost < exhaustiveCost - FLOAT_ROUND_OFF)
		{
			printf("\n\n  Nbody Extrution cost (%f) is smaller than exhaustive cost (%f). Something is wrong!\n",NbodyExtrusionCost, exhaustiveCost);
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		
		printf("\n\n  --------------------- Intermediate Run Results --------------------- ");
		
		// This is for debugging
		if(PRINT_PATHS == 1)
		{
			if(scope == 1 || scope == 2)
			{
				printf("\n\n  The exhaustive  path is     : "); 
				printPathOrder(exhaustivePath, numberOfNodes); 
				printf(" cost = %f", exhaustiveCost);
			}
			
			printf("\n\n  The nearest neighbor path is: "); 
			printPathOrder(nearestNeighborPath, numberOfNodes); 
			printf(" cost = %f", nearestNeighborCost);
			
			printf("\n\n  The Nbody extrusion path is : "); 
			printPathOrder(NbodyExtrusionPath, numberOfNodes); 
			printf(" cost = %f", NbodyExtrusionCost);
		}
		
		// Printing out the single run stats and acumulating the multiple run info to create final stats.
		// Stephen your stat collection should go here.
		if(exhaustiveCost < 0.0)
		{
			printf("\n\n  The minimum cost is unknown");
			printf("\n\n  The nearest neighbor cost is : %f", nearestNeighborCost);
			printf("\n\n  The Nbody extrusion cost is  : %f", NbodyExtrusionCost);
			
			NbodyExtrusionVSNearestNeighbor += (nearestNeighborCost - NbodyExtrusionCost)/nearestNeighborCost;
		}
		else
		{
			printf("\n\n  The minimum cost is          : %f ", exhaustiveCost);
			printf("\n\n  The nearest neighbor cost is : %f the precent error = %f", nearestNeighborCost, 100.0*(nearestNeighborCost - exhaustiveCost)/exhaustiveCost);
			printf("\n\n  The Nbody extrusion cost is  : %f the precent error = %f", NbodyExtrusionCost, 100.0*(NbodyExtrusionCost - exhaustiveCost)/exhaustiveCost);
			
			totalPercentErrorNearestNeighbor += 100.0*(nearestNeighborCost - exhaustiveCost)/exhaustiveCost;
			totalPercentErrorNbodyExtrusion  += 100.0*(NbodyExtrusionCost  - exhaustiveCost)/exhaustiveCost;
			
			NbodyExtrusionVSNearestNeighbor += (nearestNeighborCost - NbodyExtrusionCost)/nearestNeighborCost;
		}
		
		if(scope == 2 && PRINT_RAW_DATA_FILE == 1)
		{
			fprintf(RawDataFile, "  %d, %d, %f, %f, %f\n", i+1, numberOfNodes, exhaustiveCost, nearestNeighborCost, NbodyExtrusionCost);
		}
		if(scope == 4 && PRINT_RAW_DATA_FILE == 1)
		{
			fprintf(RawDataFile, "  %d, %d, %d, %f, %f, %f\n", i+1, rows, columns, exhaustiveCost, nearestNeighborCost, NbodyExtrusionCost);
		}
	
		drawFInalPicture(pos, exhaustivePath, nearestNeighborPath, NbodyExtrusionPath, scope, numberOfNodes);
		
		free(pos);
		free(exhaustivePath);
		free(nearestNeighborPath);
		free(NbodyExtrusionPath);
		free(posNbody);
		free(velNbody);
		free(accNbody);
		free(massNbody);
	}
	
	printf("\n\n\n  $$$$$$$$$$$$$$$$$$$$$$$$$ Final results $$$$$$$$$$$$$$$$$$$$$$$$$$$$$");
	
	// Printing out the final acumulated stats.
	// Stephen your stat final stats should go here.
	if(exhaustiveCost < 0.0)
	{
		printf("\n\n  The average value of the nearest neighbor method was %f on %d run(s).", totalNearestNeighborCost/numberOfRuns, numberOfRuns);
		printf("\n\n  The average value of the Nbody extrution method was %f on %d run(s).", totalNbodyExtrusionCost/numberOfRuns, numberOfRuns);
	}
	else
	{
		printf("\n\n  The average percent error of the nearest neighbor method was %f on %d runs.", totalPercentErrorNearestNeighbor/(float)numberOfRuns, numberOfRuns);
		printf("\n  The average percent error of the Nbody extrution method was %f on %d runs", totalPercentErrorNbodyExtrusion/(float)numberOfRuns, numberOfRuns);
	}
	
	NbodyExtrusionVSNearestNeighbor = 100.0*NbodyExtrusionVSNearestNeighbor/(float)numberOfRuns;
	if(NbodyExtrusionVSNearestNeighbor >= 0)
	{
		printf("\n\n  The Nbody ectrusion method was on average %f percent better than the nearest neighbor method on %d run(s).", NbodyExtrusionVSNearestNeighbor, numberOfRuns);
	}
	else
	{
		printf("\n\n  The Nbody ectrusion method was on average %f percent worse than the nearest neighbor method on %d run(s).", -NbodyExtrusionVSNearestNeighbor, numberOfRuns);
	}
	
	if(scope == 2 || scope == 4 && PRINT_RAW_DATA_FILE == 1)
	{
		fclose(RawDataFile);
	}
	
	printf("\n\nDone\n");
	while(1);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitWindowSize(X_WINDOW,Y_WINDOW);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Traveling Salesman Problem");
	glutDisplayFunc(control);
	glutMainLoop();
}



    

