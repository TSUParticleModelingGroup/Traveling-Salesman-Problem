#include "hip/hip_runtime.h"
//Maybe add a random pertibation
//I don't like that the wall strength is just an arbitrary large number
//work on pointer in exact
//Move the point off (0,0) before running the extrution program
//Move wall by a set number because the wall will not be a fixed distance apart
//
// Clean up extrution
// 
// Think about setting atraction as gravity and repultion as linear. WIll need epsilon to remove sengularity
//
// ** Here I am removing wall pressure and creating stress to move the wall (sum of stress divided by 0.5*n*(n-1)).
// In this version I am going to try and add anealing

//nvcc TravelingSalesManComparison2-26-20.cu -o temp -lglut -lGL -lm -use_fast_math
#include <GL/glut.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// General constants
#define PI 3.14159265359
#define BIG_NUMBER 1000000.0
#define SMALL_NUMBER 0.0000001

// GPU settings
#define BLOCK 256

// OpenGL drawing settings
#define X_WINDOW 800
#define Y_WINDOW 800
#define X_MAX (1.1)
#define X_MIN (-1.1)
#define Y_MAX (1.1)
#define Y_MIN (-1.1)

// Error cheaking settings
#define DRAW_EXHAUSTIVE_PATH 1
#define DRAW_NEAREST_NEIGHBOR_PATH -1
#define DRAW_NBODY_EXTRUSION_PATH 1
#define DELAY_TO_RECORD -1
#define PRINT_EDGE_COST -1
#define PRINT_EXHAUSTIVE_PATHS -1
#define PRINT_PATHS -1
#define PRINT_RAW_DATA_FILE 1

// Node creation settings
#define RANDOM_FILL_SHAPE_CIRCLE -1
#define RANDOM_FILL_SHAPE_SQUARE 1
#define NODES_TOO_CLOSE 0.000001

// General N-body settings
#define TIME_STEP_SIZE 0.002
#define DAMP 20.0 
#define MASS 80.0
#define STEPS_BETWEEN_VIEWING 100

// Force function settings
#define FORCE_FUNTION_TYPE 1 // 0 for LJ-HPQ. 1 for LJ-MPQ. 2 Linear. 3 for billards.

#define H 1.0
#define P 7.0 
#define Q 11.5  // P < Q 
#define M -0.04

#define SLOPE_REPULSION 100.0
#define	MAG_ATRACTION 10.0 //5.0
#define FORCE_CUTOFF 10.0 //2.0

#define NODE_RADIUS 0.5

// Wall move general
#define STARTING_POINT_FOR_NUMBER_OF_MOVES 1000
#define RADIUS_STEP_SIZE 0.001
#define TIME_BETWEEN_WALL_MOVES 1.0
#define WALL_STRENGTH 200000.0

// Wall move type and selection settings
#define WALL_MOVE_SCOPE 1 // 0 for nothing. 1 for pressure. 2 for average stress. 3 for average absolute stress. 4 for average positive stress. 5 for max stress

#define LOWER_PRESSURE_LIMIT 20
#define UPPER_PRESSURE_LIMIT 1000

#define LOWER_AVERAGE_STRESS_LIMIT 10
#define UPPER_AVERAGE_STRESS_LIMIT 30

#define LOWER_AVERAGE_ABSOLUTE_STRESS_LIMIT 10
#define UPPER_AVERAGE_ABSOLUTE_STRESS_LIMIT 30

#define LOWER_AVERAGE_POSITIVE_STRESS_LIMIT 10
#define UPPER_AVERAGE_POSITIVE_STRESS_LIMIT 30

#define LOWER_MAX_POSITIVE_STRESS_FROM_OUTER_WALL_LIMIT 30
#define UPPER_MAX_POSITIVE_STRESS_FROM_OUTER_WALL_LIMIT 3000

// Annealing settings
#define NUMBER_OF_ANEALINGS  0
#define ANNEALING_TIME  50.0

// Globals
FILE *RawDataFile;
FILE *StatsFile;

//Function prototypes
float x_machine_to_x_screen(int x);
float y_machine_to_y_screen(int y);
float x_machine_to_x_world(int x);
float y_machine_to_y_world(int y);
float x_world_to_x_screen(float x);
float y_world_to_y_screen(float y);
void openRawDataFile(int scope, int numberOfRuns);
void placeNodesRandom(float4 *node, unsigned int srandSeed, int scope, int n);
int checkNodes(float4 *node, int n);
void placeNodesGrid(float4 *node, int rows, int columns);
double setAverageSeperationToOne(float4 *node, int numberOfNodes);
double setMinimumSeperationToOne(float4 *node, int numberOfNodes);
float4 setGeometricCenterToZero(float4 *node, int n);
void getNumberOfNodesFromNodeFile(int *numberOfNodes, double *exhaustiveCost, char *nodeFileName);
void placeNodesFromAFile(float4 *node, int *numberOfNodes, char *nodeFileName);
void getNumberOfNodesFromASelfDefinedFunction(int *numberOfNodes);
void placeNodesFromASelfDefinedFunction(float4 *node, int numberOfNodes);
void printEdgeCosts(float4 *node, int n, double nodeAdjustmentFactor);
int factorial(int n);
void printPathOrder(int* path, int n);
double getPathCost(int *path, float4 *node, int type, int n);
void swap(int *path, int i, int j);
void heappermute(int* path, int m, float4 *node, int* exhaustivePath, double* minCost, int n);
double exhaustiveTSP(float4 *node, int* exhaustivePath, int n);
double nearestNeighborTSP(float4 *node, int* path, int n);
void setNbodyInitailConditions(float4 *node, float4 *pos, float4 *vel, float* mass, int n);
void drawPoints(float4 *pos, int n);
void drawNbodyExtrusion(float4 *pos, float innerRadius, float outerRadius, int innerWallDirection, int outerWallDirection, int n);
void getPathNbody(float4 *pos, int* path, int n);
double findMinimumSeperation(float4 *pos, int n);
int findWallMoveDirections(float4 *node, float4 *pos, int n, int scope, float outerRadius, int *innerWallDirection, int *outerWallDirection);
double NbodyExtrusionTSP(float4 *node, float4 *pos, float4 *vel, float4 *acc, float* mass, int* path, int n);
void drawFInalPicture(float4 *node, int *pathA, int *pathB, int *pathC, int scope, int n);
void getInputFromUser(int* scope, int* numberOfNodes, int* numberOfRuns, int* maxNumberOfRows, int* maxNumberOfColumns, unsigned int* srandSeed, char *nodeFileName);
void control();

float x_machine_to_x_screen(int x)
{
	return( (2.0*x)/X_WINDOW-1.0 );
}

float y_machine_to_y_screen(int y)
{
	return( -(2.0*y)/Y_WINDOW+1.0 );
}

/*	Takes machine x and y which start in the upper left corner and go from zero to X_WINDOW
	left to right and form zero to Y_WINDOW top to bottom and transslates this into world
	points which are a X_MIN to X_MAX, Y_MIN to Y_MAX window.
*/
float x_machine_to_x_world(int x)
{
	float range;
	range = X_MAX - X_MIN;
	return( (range/X_WINDOW)*x + X_MIN );
}

float y_machine_to_y_world(int y)
{
	float range;
	range = Y_MAX - Y_MIN;
	return(-((range/Y_WINDOW)*y - Y_MAX));
}

/*	Take world  points to screen points
*/
float x_world_to_x_screen(float x)
{
	float range;
	range = X_MAX - X_MIN;
	return( -1.0 + 2.0*(x - X_MIN)/range );
}

float y_world_to_y_screen(float y)
{
	float range;
	range = Y_MAX - Y_MIN;
	return( -1.0 + 2.0*(y - Y_MIN)/range );
}

void openRawDataFile(int scope, int numberOfRuns)
{
	char tagName[50];
	char fileName[256];
	
	strcpy(tagName,"");
	if(scope == 2 || scope == 6)
	{
		strcat(tagName,"Random_TSP_Raw_Data_CSV");
	}
	if(scope == 4)
	{
		strcat(tagName,"Grid_TSP_Raw_Data_CSV");
	}
	
	snprintf(fileName, 256, "%s", tagName);
	
	RawDataFile = fopen(fileName, "wb");
  	
	fprintf(RawDataFile, "  Number of runs %d\n",numberOfRuns);
	
	if(scope == 2 || scope == 6)
	{
		fprintf(RawDataFile, "  Run  Nodes  Exact  NNeighbor  NBody\n\n");
	}
	if(scope == 4)
	{
		fprintf(RawDataFile, "  Run  Rows  Columns  Exact  NNeighbor  NBody\n\n");
	}
}

void placeNodesRandom(float4 *node, unsigned int srandSeed, int scope, int n)
{
	time_t t;
	double xStart, xStop, yStart, yStop, radius, maxRadius;
	double mag, seperation;
	int repeatedNodeTest;
	
	xStart = -sqrt(n)/2.0;
	xStop  = sqrt(n)/2.0;
	yStart = -sqrt(n)/2.0;
	yStop  = sqrt(n)/2.0;
	
	maxRadius = sqrt(n/2.0);

	srand((unsigned) time(&t));

	if(scope == 1 || scope == 5) srand(srandSeed);
	else srand((unsigned) time(&t));

	if(RANDOM_FILL_SHAPE_SQUARE == 1)
	{
		for(int i = 0; i < n; i++)
		{
			repeatedNodeTest = 0;
			while(repeatedNodeTest == 0)
			{
				node[i].x = (double)rand()/((double)RAND_MAX)*(xStop - xStart) + xStart;
				node[i].y = (double)rand()/((double)RAND_MAX)*(yStop - yStart) + yStart;
				repeatedNodeTest = 1;
				for(int j = 0; j < i; j++)
				{
					seperation = sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));
					if(seperation < NODES_TOO_CLOSE)
					{
						repeatedNodeTest = 0;
						break;
					}
				}
			}
		}
	}
	
	if(RANDOM_FILL_SHAPE_CIRCLE == 1)
	{
		for(int i = 0; i < n; i++)
		{
			repeatedNodeTest = 0;
			while(repeatedNodeTest == 0)
			{
				node[i].x = (double)rand()/((double)RAND_MAX)*(xStop - xStart) + xStart;
				node[i].y = (double)rand()/((double)RAND_MAX)*(yStop - yStart) + yStart;
				mag = sqrt(node[i].x*node[i].x + node[i].y*node[i].y);
				radius = ((double)rand()/(double)RAND_MAX)*maxRadius;
				node[i].x *= radius/mag;
				node[i].y *= radius/mag;
				repeatedNodeTest = 1;
				for(int j = 0; j < i; j++)
				{
					seperation = sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));  
					if(seperation < NODES_TOO_CLOSE)
					{
						repeatedNodeTest = 0;
						break;
					}
				}
			}
		}
	}
	//printf("\n %f  %f  %f  %f", node[63].x, node[63].y, node[2322].x, node[2322].y);
}

int checkNodes(float4 *node, int n)
{
	double seperation;
	
	for(int i = 0; i < (n - 1); i++)
	{
		for(int j = (i + 1); j < n; j++)
		{
			seperation = sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));
			if(seperation < NODES_TOO_CLOSE)
			{
			//printf("\n i = %d j = %d", i,j);
				return(-1);
			}
		}
	}
	return(1);
}

void placeNodesGrid(float4 *node, int rows, int columns)
{
	int i,j,k;
	double dx, dy;
	double xStart, yStart;
	
	xStart = 0.0;
	yStart = 0.0;
	dx = 1.0;
	dy = 1.0;
	
	k = 0;
	for(i = 0; i < columns; i++)
	{
		for(j = 0; j < rows; j++)
		{
			node[k].x = xStart + dx*i;
			node[k].y = yStart + dy*j;
			k++;
		}
	}
}

void getNumberOfNodesFromNodeFile(int *numberOfNodes, double *exhaustiveCost, char *nodeFileName)
{
	FILE *nodeFile;
	nodeFile = fopen(nodeFileName, "rb");
	fscanf(nodeFile,"%d", numberOfNodes);
	fscanf(nodeFile,"%lf", exhaustiveCost);
	fclose(nodeFile);
}

void placeNodesFromAFile(float4 *node, int *numberOfNodes, char *nodeFileName)
{
	FILE *nodeFile;
	double exhaustiveCost ;
	nodeFile = fopen(nodeFileName, "rb");
	fscanf(nodeFile,"%d", numberOfNodes);
	fscanf(nodeFile,"%lf", &exhaustiveCost);
	
	//printf("\n\n numberOfNodes = %d, exhaustiveCost = %lf\n\n", *numberOfNodes, exhaustiveCost);
	
	for(int i = 0; i < *numberOfNodes; i++)
	{
		//fscanf(nodeFile,"%d %f %f", &nodeNumber, &node[i].x, &node[i].y );
		fscanf(nodeFile,"%f %f", &node[i].x, &node[i].y );
		//printf("\n\n node number= %d, %f, %f\n\n", i, node[i].x, node[i].y);
	}
	
	fclose(nodeFile);
}

void getNumberOfNodesFromASelfDefinedFunction(int *numberOfNodes)
{
	*numberOfNodes = 12*16;
}

void placeNodesFromASelfDefinedFunction(float4 *node, int numberOfNodes)
{
	int i, j, k, count;
	float delta, shiftX, shiftY;
	
	delta = 8.4;
	count = 0;
	for(j = 0; j <2; j++)
	{
		shiftX = -4.2 + j*delta;
		for(k = 0; k <2; k++)
		{
			shiftY = -4.2 + k*delta;
			for(i = 0; i < 12; i++)
			{
				node[count].x = 2.0*cos(i*2.0*PI/(12.0)) + 2.1 + shiftX;
				node[count].y = 2.0*sin(i*2.0*PI/(12.0)) + 2.1 + shiftY;
				count++;
			}
			for(i = 0; i < 12; i++)
			{
				node[count].x = 2.0*cos(i*2.0*PI/(12.0)) - 2.1 + shiftX;
				node[count].y = 2.0*sin(i*2.0*PI/(12.0)) + 2.1 + shiftY;
				count++;
			}
			for(i = 0; i < 12; i++)
			{
				node[count].x = 2.0*cos(i*2.0*PI/(12.0)) - 2.1 + shiftX;
				node[count].y = 2.0*sin(i*2.0*PI/(12.0)) - 2.1 + shiftY;
				count++;
			}
			for(i = 0; i < 12; i++)
			{
				node[count].x = 2.0*cos(i*2.0*PI/(12.0)) + 2.1 + shiftX;
				node[count].y = 2.0*sin(i*2.0*PI/(12.0)) - 2.1 + shiftY;
				count++;
			}
		}
	}
	
	/*
	for(i = 0; i < 12; i++)
	{
		node[count].x = 2.0*cos(i*2.0*PI/(12.0)) + 2.1 + 8.1;
		node[count].y = 2.0*sin(i*2.0*PI/(12.0)) + 2.1;
		count++;
	}
	for(i = 0; i < 12; i++)
	{
		node[count].x = 2.0*cos(i*2.0*PI/(12.0)) - 2.1 + 8.1;
		node[count].y = 2.0*sin(i*2.0*PI/(12.0)) + 2.1;
		count++;
	}
	for(i = 0; i < 12; i++)
	{
		node[count].x = 2.0*cos(i*2.0*PI/(12.0)) - 2.1 + 8.1;
		node[count].y = 2.0*sin(i*2.0*PI/(12.0)) - 2.1;
		count++;
	}
	for(i = 0; i < 12; i++)
	{
		node[count].x = 2.0*cos(i*2.0*PI/(12.0)) + 2.1 + 8.1;
		node[count].y = 2.0*sin(i*2.0*PI/(12.0)) - 2.1;
		count++;
	}
	*/
}

//This function adjustes the nodes so that the average seperation is 1.0
double setAverageSeperationToOne(float4 *node, int numberOfNodes)
{
	double nodeAdjustmentFactor;
	double sum;
	int numberOfEdges;
	double dx,dy;
	int i,j;
	
	sum = 0.0;
	for(i = 0; i < numberOfNodes; i++)
	{
		for(j = i + 1; j < numberOfNodes; j++)
		{
			dx = node[i].x-node[j].x;
			dy = node[i].y-node[j].y;
			sum += sqrt(dx*dx + dy*dy);
		}
	}
	
	numberOfEdges = ((numberOfNodes)*(numberOfNodes - 1))/2;
	nodeAdjustmentFactor = sum/numberOfEdges;
	
	for(int i = 0; i < numberOfNodes; i++)
	{
		node[i].x = node[i].x/nodeAdjustmentFactor;
		node[i].y = node[i].y/nodeAdjustmentFactor;
	}
	
	return(nodeAdjustmentFactor);
}

//This function adjustes the nodes so that the minimum seperation is 1.0
double setMinimumSeperationToOne(float4 *node, int numberOfNodes)
{
	double nodeAdjustmentFactor;
	double minimum;
	double dx,dy, d;
	int i,j;
	
	minimum = BIG_NUMBER;
	for(i = 0; i < numberOfNodes; i++)
	{
		for(j = i + 1; j < numberOfNodes; j++)
		{
			dx = node[i].x-node[j].x;
			dy = node[i].y-node[j].y;
			d = sqrt(dx*dx + dy*dy);
			if(d < minimum) minimum = d;
		}
	}
	
	nodeAdjustmentFactor = minimum;
	
	for(int i = 0; i < numberOfNodes; i++)
	{
		node[i].x = node[i].x/nodeAdjustmentFactor;
		node[i].y = node[i].y/nodeAdjustmentFactor;
	}
	
	return(nodeAdjustmentFactor);
}

float4 setGeometricCenterToZero(float4 *node, int n)
{
	float4 geometricCenter;
	
	geometricCenter.x = 0.0;
	geometricCenter.y = 0.0;
	
	for(int i = 0; i < n; i++)
	{
		geometricCenter.x += node[i].x;
		geometricCenter.y += node[i].y;
	}
	
	geometricCenter.x /= (float)n;
	geometricCenter.y /= (float)n;
	
	for(int i = 0; i < n; i++)
	{
		node[i].x -= geometricCenter.x;
		node[i].y -= geometricCenter.y;
	}
	return(geometricCenter);
}

void moveAnyNodeOffDeadCenter(float4 *pos, int n)
{
	int i;

	for(i = 0; i < n; i++)
	{
		if( sqrt(pos[i].x*pos[i].x + pos[i].y*pos[i].y) < 0.001) 
		{
			pos[i].x = 0.001;
			pos[i].y = 0.001;
		}
	}
}

double findMinimumSeperation(float4 *pos, int n)
{
	int i,j;
	double min;
	double temp;
	
	min = BIG_NUMBER;
	
	for(i = 0; i < n; i++)
	{
		for(j = i + 1; j < n; j++)
		{
			temp = sqrt((pos[i].x-pos[j].x)*(pos[i].x-pos[j].x) + (pos[i].y-pos[j].y)*(pos[i].y-pos[j].y));
			if( temp < min) 
			{
				min = temp;
			}
		}
	}
	if(min < SMALL_NUMBER)
	{
		printf("\n  You have at least two nodes in the same location\n");
		printf("\n 	Minimum seperation = %f \n", min);
		printf("\n  Good Bye\n");
		exit(0);
	}
	return(min);
}

double findDistanceToOuterMostElement(float4 *element, int numberOfElements)
{
	double temp;
	double distanceToOutermostElement = 0.0;
	
	for(int i = 0; i < numberOfElements; i++)
	{
		temp = sqrt(element[i].x*element[i].x + element[i].y*element[i].y);
		if(temp > distanceToOutermostElement)
		{
			distanceToOutermostElement = temp;
		}
	}
	return(distanceToOutermostElement);
}

double findForceCPU(float4 node1, float4 node2, float4 pos1, float4 pos2)
{
	double naturalLength, actualLength, deltaLength, forceMag;
	
	naturalLength = sqrt((node1.x - node2.x)*(node1.x - node2.x) + (node1.y - node2.y)*(node1.y - node2.y));
	actualLength  = sqrt((pos1.x - pos2.x)*(pos1.x - pos2.x) + (pos1.y - pos2.y)*(pos1.y - pos2.y));
	deltaLength = actualLength - naturalLength;
	
	if(deltaLength <= 0.0)
	{
		forceMag = -deltaLength*SLOPE_REPULSION;
	}
	else if(actualLength < FORCE_CUTOFF)
	{
		forceMag =  MAG_ATRACTION/naturalLength;
	}
	else
	{
		forceMag = 0.0;
	}
	return(forceMag);
}

int findWallMoveDirections(float4 *node, float4 *pos, int n, int scope, float outerRadius, int *innerWallDirection, int *outerWallDirection)
{
	int i,j;
	int count;
	double sum, temp, maxValue, testValue;
	float lowerLimit, upperLimit;
	
	if(scope == 0) // No adjustment inner wall just moves out.
	{
		*innerWallDirection = 1;
		*outerWallDirection = 0;
		return(1);
	}

	if(scope == 1) // Move walls by pressure on the outer wall.
	{
		lowerLimit = LOWER_PRESSURE_LIMIT;
		upperLimit = UPPER_PRESSURE_LIMIT;
		sum = 0.0;
		count = 0;
		for(i = 0; i < n; i++)
		{
			temp = sqrt(pos[i].x*pos[i].x + pos[i].y*pos[i].y) - outerRadius;
			if(0.0 < temp) 
			{
				sum += temp;
			}
		}
		testValue = sum*WALL_STRENGTH/(2.0*PI*outerRadius);
	}
	
	if(scope == 2) // Move walls by average stress on the nodes.
	{
		lowerLimit = LOWER_AVERAGE_STRESS_LIMIT;
		upperLimit = UPPER_AVERAGE_STRESS_LIMIT;
		sum = 0.0;
		count = 0;
		for(i = 0; i < n; i++)
		{
			for(j = i + 1; j < n; j++)
			{
				sum += findForceCPU(node[i], node[j], pos[i], pos[j]);
				count++;
			}
		}
		testValue = sum/count;
	}
	
	if(scope == 3) // Move walls by absolute average stress.
	{
		lowerLimit = LOWER_AVERAGE_ABSOLUTE_STRESS_LIMIT;
		upperLimit = UPPER_AVERAGE_ABSOLUTE_STRESS_LIMIT;
		sum = 0.0;
		count = 0;
		for(i = 0; i < n; i++)
		{
			for(j = i + 1; j < n; j++)
			{
				sum += abs(findForceCPU(node[i], node[j], pos[i], pos[j]));
				count++;
			}
		}
		testValue = sum/count;
	}
	
	if(scope == 4)
	{
		lowerLimit = LOWER_AVERAGE_POSITIVE_STRESS_LIMIT;
		upperLimit = UPPER_AVERAGE_POSITIVE_STRESS_LIMIT;
		sum = 0.0;
		count = 0;
		for(i = 0; i < n; i++)
		{
			for(j = i + 1; j < n; j++)
			{
				temp = findForceCPU(node[i], node[j], pos[i], pos[j]);
				if(0.0 < temp) 
				{
					sum += temp;
					count++;
				}
			}
		}
		testValue = sum/count;
	}
	
	if(scope == 5)
	{
		lowerLimit = LOWER_MAX_POSITIVE_STRESS_FROM_OUTER_WALL_LIMIT;
		upperLimit = UPPER_MAX_POSITIVE_STRESS_FROM_OUTER_WALL_LIMIT;
		maxValue = 0.0;
		for(i = 0; i < n; i++)
		{
			if(0.0 <= sqrt(pos[i].x*pos[i].x + pos[i].y*pos[i].y) - outerRadius) 
			{
				for(j = i + 1; j < n; j++)
				{
					temp = findForceCPU(node[i], node[j], pos[i], pos[j]);
					if(maxValue < temp) 
					{
						maxValue = temp;
					}
				}
			}
		}
		testValue = maxValue;
	}
	
	if(testValue < lowerLimit)
	{
		*innerWallDirection = 0;
		*outerWallDirection = -1;
	}
	else if(testValue < upperLimit)
	{
		*innerWallDirection = 1;
		*outerWallDirection = 0;
	}
	else
	{
		*innerWallDirection = 0;
		*outerWallDirection = 1;
	}
		
	return(1);
}

void printEdgeCosts(float4 *node, int n, float nodeAdjustmentFactor)
{
	double temp;
	for(int i = 0; i < n; i++)
	{
		for(int j = i + 1; j < n; j++)
		{	
			temp = sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y))*nodeAdjustmentFactor;
			printf("edge cost [%d, %d] = %f\n", i, j, temp);
		}
	}
}

int factorial(int n)
{
	int outPut = n;
	
	for(int i = n-1; i > 0; i--)
	{
		outPut *= i;	
	}
	return(outPut);
}

void printPathOrder(int* path, int n)
{
	printf("  ");
	for(int i = 0; i < n-1; i++)
	{
		printf("%d->", path[i]);	
	}
	printf("%d", path[n-1]);
}

double getPathCost(int *path, float4 *node, int type, int n)
{
	double cost;
	int i, j, k;
	
	//Checking path validaty 
	for(i = 0; i < n; i++)
	{
		if(path[i] < 0 || (n-1) < path[i])
		{
			printf("\n\n  Error -> Path out of range! Type = %d", type);
			printf("\n  path[%d] = %d\n\n", i, path[i]);
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		
		for(j = 0; j < i; j++)
		{
			if(path[i] == path[j])
			{
				printf("\n\n Error -> Path has a repeated index! Type = %d\n", type);
				printPathOrder(path, n);
				printf("\n\n");
				printf("\n\n  Good Bye.  \n\n");
				exit(0);
			}
		}
	}
	
	cost = 0.0;
	for(k = 0; k < n-1; k++)
	{
		i = path[k];
		j = path[k+1];
		cost += sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));
	}
	i = path[n-1];
	j = path[0];
	cost += sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));
	
	return(cost);
}

void swap(int *path, int i, int j)
{
	int temp;
	temp = path[i];
	path[i] = path[j];
	path[j] = temp;
}

void heappermute(int *path, int m, float4 *node, int *exhaustivePath, double *minCost, int n) 
{
	int i;
	double pathCost;
	int* pathPlus = (int*)malloc(n*sizeof(int));

	if (m == 1) 
	{
		pathPlus[0] = 0;
		for(i = 1; i < n; i++)
		{
			pathPlus[i] = path[i-1];	
		}
		
		pathCost = getPathCost(pathPlus, node, 1, n);
		
		if(PRINT_EXHAUSTIVE_PATHS == 1)
		{
			printf("\n");
			printPathOrder(pathPlus, n);
			printf(" cost = %f", pathCost);
		}
		
		if(pathCost < minCost[0])
		{
			minCost[0] = pathCost;
			for(i = 0; i < n; i++)
			{
				exhaustivePath[i] = pathPlus[i];	
			}
		}
    	}
	else 
	{
		for (i = 0; i < m; i++) 
		{
			heappermute(path, m-1, node, exhaustivePath, minCost, n);
			if (m % 2 == 1) 
			{
				swap(path, 0, m-1);
			}
			else 
			{
				swap(path, i, m-1);
			}
		}
	}
	free(pathPlus);
}

double exhaustiveTSP(float4 *node, int* exhaustivePath, int n)
{
	double cost[1];
	int* path = (int*)malloc((n-1)*sizeof(int));
	
	exhaustivePath[0] = 0;
	for(int i = 1; i < n; i++)
	{
		exhaustivePath[i] = i;
		path[i-1] = i;	
	}
	cost[0] = getPathCost(exhaustivePath, node, 1, n);
	
	heappermute(path, n-1, node, exhaustivePath, cost, n);
	free(path);
	return(cost[0]);
}

double nearestNeighborTSP(float4 *node, int* path, int n)
{
	int i, j, k, nextNode, nodeFound;
	double minCost, pathCost, edgeCost, maxEdgeCost;
	int* used = (int*)malloc(n*sizeof(int));
	
	maxEdgeCost = 0.0;
	for(i = 0; i < n; i++)
	{
		for(j = 0; j < n; j++)
		{
			edgeCost = sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));
			if(edgeCost > maxEdgeCost) 
			{
				maxEdgeCost = edgeCost;
			}	
		}	
	}
	maxEdgeCost += 1.0;
	
	for(i = 0; i < n; i++)
	{
		used[i] = -1;	
	}
	
	path[0] = 0;
	used[0] = 1;
	
	k = 0;
	
	minCost = maxEdgeCost;
	while(k < n-1)
	{
		nodeFound = 0;
		for(j = 0; j < n; j++)
		{
			i = path[k];
			edgeCost = sqrt((node[i].x-node[j].x)*(node[i].x-node[j].x) + (node[i].y-node[j].y)*(node[i].y-node[j].y));
			if(edgeCost <= minCost && used[j] == -1)
			{
				minCost = edgeCost;
				nextNode = j;
				nodeFound = 1;
			}	
		}
		if(nodeFound == 0)
		{
			printf("\n\n  There was a problem in the nearest neighbor function. No next node was found.\n\n");
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		nodeFound = 0;
		
		k++;
		path[k] = nextNode;
		used[nextNode] = 1;
		minCost = maxEdgeCost;
	}
	
	pathCost = getPathCost(path, node, 2, n);
	free(used);
	return(pathCost);
}

void setNbodyInitailConditions(float4 *node, float4 *pos, float4 *vel, float* mass, int n)
{
	int i;

	for(i = 0; i < n; i++)
	{
		pos[i].x = node[i].x;
		pos[i].y = node[i].y;
		
		vel[i].x = 0.0;
		vel[i].y = 0.0;
		
		mass[i] = MASS;
	}
}

__device__ float getBodyBodyForceMag(float edgeLength, float d)
{
	float forceMag;
	float c, h;
	
	if(FORCE_FUNTION_TYPE == 0)  // Lenard Jones type force
	{
		forceMag = -H*powf(edgeLength, Q - P)/powf(d, Q) + H/powf(d, P);
	}
	if(FORCE_FUNTION_TYPE == 1)  // Lenard Jones type force
	{
		//d = dist(shPos[i], pos) + 0.01;

		h = M*(powf(powf(Q/P, 1/(Q-P))*edgeLength, P))/(1 - P/Q);
		
		c = powf(edgeLength/d, Q - P);
		forceMag = (c-1)*h/powf(d, P);
	}
	else if(FORCE_FUNTION_TYPE == 2)  // Linear function
	{
		if(d <= edgeLength)
		{
			forceMag = -(edgeLength - d)*SLOPE_REPULSION;

		}
		else if(edgeLength < d && d < FORCE_CUTOFF)
		{
			forceMag =  MAG_ATRACTION/edgeLength;
		}
		else
		{
			forceMag = 0.0;
		}
	}
	else if(FORCE_FUNTION_TYPE == 3)  // Billards function
	{
		if(d <= NODE_RADIUS)
		{
			forceMag = -edgeLength*(NODE_RADIUS - d);

		}
		else
		{
			forceMag = 0.0;
		}
	}
	else
	{
		forceMag = 0.0;
	}
	
	return(forceMag);
	
	
}

__global__ void accelerationsNbody(float4 *node, float4 *pos, float4 *vel, float4 *acc, float *mass, float innerRadius, float outerRadius, int n)
{
	int j,ii;
    float3 forceSum;
    float4 nodeMe, posMe;
    float dx, dy, d, edgeLength; 
    float radius, forceMag;
    __shared__ float4 shnode[BLOCK], shPos[BLOCK];
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    
    forceSum.x = 0.0;
	forceSum.y = 0.0;
	
	nodeMe.x = node[id].x;
	nodeMe.y = node[id].y;
	posMe.x = pos[id].x;
	posMe.y = pos[id].y;
		    
    for(j=0; j < gridDim.x; j++)
    {
    	if(threadIdx.x + blockDim.x*j < n)
    	{
    		shPos[threadIdx.x] = pos[threadIdx.x + blockDim.x*j];
    		shnode[threadIdx.x] = node[threadIdx.x + blockDim.x*j];
    	}
    	__syncthreads();
   
		#pragma unroll 32
        for(int i = 0; i < blockDim.x; i++)	
        {
        	ii = i + blockDim.x*j;
		    if(ii != id && ii < n) 
		    {
				dx = shPos[i].x - posMe.x;
				dy = shPos[i].y - posMe.y;
				d = sqrtf(dx*dx + dy*dy);
				
				edgeLength = sqrtf((shnode[i].x - nodeMe.x)*(shnode[i].x - nodeMe.x) + (shnode[i].y - nodeMe.y)*(shnode[i].y - nodeMe.y));
				
				forceMag = getBodyBodyForceMag(edgeLength, d);
				forceSum.x += forceMag*dx/d;
				forceSum.y += forceMag*dy/d;
		    }
		}
	}
	
	if(id < n)
	{
		// Forces between node and the walls
		dx = posMe.x;
		dy = posMe.y; 
		radius = sqrtf(dx*dx + dy*dy);
	
		if(radius < innerRadius) // Inside inner wall
		{
			forceMag = WALL_STRENGTH*(innerRadius - radius);
			forceSum.x += forceMag*dx/radius;
			forceSum.y += forceMag*dy/radius;
		}
		else if(radius > outerRadius) // Outside outer wall
		{
			forceMag = WALL_STRENGTH*(outerRadius - radius);
			forceSum.x += forceMag*dx/radius;
			forceSum.y += forceMag*dy/radius;
		}
		
		// Adding on damping force.
		forceSum.x += -DAMP*vel[id].x;
		forceSum.y += -DAMP*vel[id].y;
		
		// Creating the accelerations.
	    acc[id].x = forceSum.x/mass[id];
	    acc[id].y = forceSum.y/mass[id];
    }
}

__global__ void moveNbody(float4 *pos, float4 *vel, float4 *acc, float dt, int n)
{
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    if(id < n)
    {
	    vel[id].x += acc[id].x*dt;
		vel[id].y += acc[id].y*dt;
		
		pos[id].x  += vel[id].x*dt;
		pos[id].y  += vel[id].y*dt;
    }
}

void getPathNbody(float4 *pos, int* path, int n)
{
	int i;
	double minValue;
	double *angle = (double*)malloc(n*sizeof(double));
	int *used = (int*)malloc(n*sizeof(int));
	
	for(i = 0; i < n; i++)
	{
		if(pos[i].x == 0 && pos[i].y == 0)
		{
			angle[i] = 0.0;
		}
		else if(pos[i].x >= 0 && pos[i].y >= 0)
		{
			if(pos[i].x == 0) angle[i] = 90.0;
			else angle[i] = atan(pos[i].y/pos[i].x)*180.0/PI;
		}
		else if(pos[i].x < 0 && pos[i].y >= 0)
		{
			angle[i] = 180.0 - atan(pos[i].y/(-pos[i].x))*180.0/PI;
		}
		else if(pos[i].x <= 0 && pos[i].y < 0)
		{
			if(pos[i].x == 0) angle[i] = 270.0;
			else angle[i] = 180.0 + atan(pos[i].y/pos[i].x)*180.0/PI;
		}
		else
		{
			angle[i] = 360.0 - atan(-pos[i].y/pos[i].x)*180.0/PI;
		}
	}
	
	for(i = 0; i < n; i++)
	{
		used[i] = 0;
	}
	
	for(int k = 0; k < n; k++)
	{
		minValue = 400.0;
		for(i = 0; i < n; i++)
		{
			if(angle[i] < minValue && used[i] == 0)
			{
				minValue = angle[i];
				path[k] = i;
			}
		}
		used[path[k]] = 1;
		//printf("path[%d] = %d\n", k, path[k]);
	}
	
	free(angle);
	free(used);
}

double NbodyExtrusionTSP(float4 *node, float4 *pos, float4 *vel, float4 *acc, float* mass, int* path, int n)
{
	int draw_count;
	int innerWallDirection, outerWallDirection;
	int annealingCount;
	double dr;
	float dt = TIME_STEP_SIZE;
	double pathCost;
	double time;
	float innerRadius, outerRadius;
	double stopSeperation;
	
	dim3 block, grid;
	float4 *posGPU, *velGPU, *accGPU; 
	float *massGPU;
	float4 *nodeGPU;
	
	// Setting up GPU parrellel structure.
	block.x = BLOCK;
	block.y = 1;
	block.z = 1;
	
	grid.x = (n-1)/block.x + 1;
	grid.y = 1;
	grid.z = 1;
	
	// Allocating memory.
	hipMalloc( (void**)&nodeGPU, n *sizeof(float4));
	hipMalloc( (void**)&posGPU, n *sizeof(float4));
	hipMalloc( (void**)&velGPU, n *sizeof(float4));
	hipMalloc( (void**)&accGPU, n *sizeof(float4));
	hipMalloc( (void**)&massGPU, n *sizeof(float));
	
	// This is used to pause the program so you can setup to take a video of a run.
	if(DELAY_TO_RECORD == 1)
	{
		printf("\n\n  Enter a character to start\n\n"); getchar();
	}
	
	// Copying information up to the GPU.
	hipMemcpy( nodeGPU, node, n *sizeof(float4), hipMemcpyHostToDevice );
	hipMemcpy( posGPU, pos, n *sizeof(float4), hipMemcpyHostToDevice );
    hipMemcpy( velGPU, vel, n *sizeof(float4), hipMemcpyHostToDevice );
    hipMemcpy( massGPU, mass, n *sizeof(float), hipMemcpyHostToDevice );
	
	annealingCount = 0;
	while(annealingCount <= NUMBER_OF_ANEALINGS)
	{
		moveAnyNodeOffDeadCenter(pos, n);
		stopSeperation = findMinimumSeperation(pos, n)/1.0;
		innerRadius = 0.0;
		outerRadius = findDistanceToOuterMostElement(pos, n);
		drawNbodyExtrusion(pos, innerRadius, outerRadius, 0, 0, n);
		
		outerWallDirection = 0;
		innerWallDirection = 0;
		dr = outerRadius/STARTING_POINT_FOR_NUMBER_OF_MOVES;
		draw_count = 0;
	
		while(innerRadius + stopSeperation < outerRadius)
		//while(innerRadius + stopSeperation < dr)
		{
			outerRadius += dr*outerWallDirection;
			innerRadius += dr*innerWallDirection;
			time = 0.0;
			while(time < TIME_BETWEEN_WALL_MOVES)
			{		
				accelerationsNbody<<<grid, block>>>(nodeGPU, posGPU, velGPU, accGPU, massGPU, innerRadius, outerRadius, n);
				moveNbody<<<grid, block>>>(posGPU, velGPU, accGPU, dt, n);
			
				if(draw_count == STEPS_BETWEEN_VIEWING)
				{
					hipMemcpy( pos, posGPU, n *sizeof(float4), hipMemcpyDeviceToHost );
					drawNbodyExtrusion(pos, innerRadius, outerRadius, innerWallDirection, outerWallDirection, n);
					draw_count = 0;
				}
				draw_count++;
				time += dt;
			}
			hipMemcpy( pos, posGPU, n *sizeof(float4), hipMemcpyDeviceToHost );
			findWallMoveDirections(node, pos, n, WALL_MOVE_SCOPE, outerRadius, &innerWallDirection, &outerWallDirection);
			dr = outerRadius/STARTING_POINT_FOR_NUMBER_OF_MOVES;
		}
		
		if(annealingCount < NUMBER_OF_ANEALINGS)
		{
			innerRadius = 0.0;
			outerRadius = outerRadius + outerRadius*0.7;
			time = 0.0;
			while(time < ANNEALING_TIME)
			{		
				accelerationsNbody<<<grid, block>>>(nodeGPU, posGPU, velGPU, accGPU, massGPU, innerRadius, outerRadius, n);
				moveNbody<<<grid, block>>>(posGPU, velGPU, accGPU, dt, n);
			
				if(draw_count == STEPS_BETWEEN_VIEWING)
				{
					hipMemcpy( pos, posGPU, n *sizeof(float4), hipMemcpyDeviceToHost );
					drawNbodyExtrusion(pos, innerRadius, outerRadius, innerWallDirection, outerWallDirection, n);
					draw_count = 0;
				}
				draw_count++;
				time += dt;
			}
		}
		annealingCount++;
	}
	getPathNbody(pos, path, n);
	pathCost = getPathCost(path, node, 3, n);
	
	return(pathCost);
}

void drawPoints(float4 *pos, int n)
{
	int i;
	float outerRadius = findDistanceToOuterMostElement(pos, n);
	float normalizingFactor = outerRadius;
	
	glClear(GL_COLOR_BUFFER_BIT);
	
	glPointSize(5.0);
	glColor3f(1.0,0.0,0.0);
	for(i = 0; i < n; i++)
	{
		glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(pos[i].x/normalizingFactor),y_world_to_y_screen(pos[i].y/normalizingFactor));
		glEnd();

	}
	glFlush();
}

void drawNbodyExtrusion(float4 *pos, float innerRadius, float outerRadius, int innerWallDirection, int outerWallDirection, int n)
{
	int i;
	int lineAmount = 100;
	float normalizingFactor = outerRadius;
	
	outerRadius /= normalizingFactor;
	innerRadius /= normalizingFactor;

	glClear(GL_COLOR_BUFFER_BIT);
	
	GLfloat twicePi = 2.0f * PI;
	
	glLineWidth(1.0);
	if(innerWallDirection == -1) glColor3f(1.0,0.0,0.0);
	else if(innerWallDirection == 0) glColor3f(1.0,1.0,0.0);
	else glColor3f(0.0,0.0,1.0);
	glBegin(GL_LINE_LOOP);
		for(i = 0; i <= lineAmount;i++) 
		{ 
			glVertex2f(x_world_to_x_screen(innerRadius*cos(i*twicePi/lineAmount)), 
			           y_world_to_y_screen(innerRadius*sin(i*twicePi/lineAmount)));
		}
	glEnd();
	
	glLineWidth(1.0);
	if(outerWallDirection == -1) glColor3f(1.0,0.0,0.0);
	else if(outerWallDirection == 0) glColor3f(1.0,1.0,0.0);
	else glColor3f(0.0,0.0,1.0);
	glBegin(GL_LINE_LOOP);
		for(i = 0; i <= lineAmount;i++) 
		{ 
			glVertex2f(x_world_to_x_screen(outerRadius*cos(i*twicePi/lineAmount)), 
			           y_world_to_y_screen(outerRadius*sin(i*twicePi/lineAmount)));
		}
	glEnd();
	
	glPointSize(5.0);
	glColor3f(1.0,0.0,0.0);
	for(i = 0; i < n; i++)
	{
		glBegin(GL_POINTS);
		glVertex2f(x_world_to_x_screen(pos[i].x/normalizingFactor),y_world_to_y_screen(pos[i].y/normalizingFactor));
		glEnd();

	}
	
	glFlush();
}

void drawFInalPicture(float4 *node, int *pathA, int *pathB, int *pathC, int scope, int n)
{	
	int i;
	float outerRadius = findDistanceToOuterMostElement(node, n);
	float normalizingFactor = outerRadius; //((float)n)/IDEAL_NUMBER_OF_NODES;

	glClear(GL_COLOR_BUFFER_BIT);
	
	//exhuastivePath path
	if(scope == 1 || scope == 2)
	{
		if(DRAW_EXHAUSTIVE_PATH == 1)
		{
			glLineWidth(6.0);
			glColor3f(0.0,0.0,1.0);
			glBegin(GL_LINE_LOOP);
				for(i = 0; i < n; i++)
				{
					glVertex2f(x_world_to_x_screen(node[pathA[i]].x/normalizingFactor),y_world_to_y_screen(node[pathA[i]].y/normalizingFactor));
				}
			glEnd();
		}
	}
	
	//Nearest Neighbor path
	if(DRAW_NEAREST_NEIGHBOR_PATH == 1)
	{
		glLineWidth(6.0);
		glColor3f(0.0,1.0,0.0);
		glBegin(GL_LINE_LOOP);
			for(i = 0; i < n; i++)
			{
				glVertex2f(x_world_to_x_screen(node[pathB[i]].x/normalizingFactor),y_world_to_y_screen(node[pathB[i]].y/normalizingFactor));
			}
		glEnd();
	}
	
	//Nbody Extrusion path
	if(DRAW_NBODY_EXTRUSION_PATH == 1)
	{
		glLineWidth(3.0);
		glColor3f(1.0,0.0,0.0);
		glBegin(GL_LINE_LOOP);
			for(i = 0; i < n; i++)
			{
				glVertex2f(x_world_to_x_screen(node[pathC[i]].x/normalizingFactor),y_world_to_y_screen(node[pathC[i]].y/normalizingFactor));
			}
		glEnd();
	}
	
	//Placing nodes
	glPointSize(8.0);
	glColor3f(1.0,1.0,1.0);
	for(i = 0; i < n; i++)
	{
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(node[i].x/normalizingFactor),y_world_to_y_screen(node[i].y/normalizingFactor));
		glEnd();
	}
	
	//Nearest neighbor start node 
	if(DRAW_NEAREST_NEIGHBOR_PATH == 1)
	{
		glPointSize(10.0);
		glColor3f(0.0,0.0,1.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(node[pathB[0]].x/normalizingFactor),y_world_to_y_screen(node[pathB[0]].y/normalizingFactor));
		glEnd();
	}
	
	//Nbody extrution start and stop nodes
	if(DRAW_NBODY_EXTRUSION_PATH == 1)
	{
		glPointSize(10.0);
		glColor3f(0.0,1.0,0.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(node[pathC[0]].x/normalizingFactor),y_world_to_y_screen(node[pathC[0]].y/normalizingFactor));
		glEnd();
	
		glColor3f(1.0,0.0,0.0);
		glBegin(GL_POINTS);
			glVertex2f(x_world_to_x_screen(node[pathC[n-1]].x/normalizingFactor),y_world_to_y_screen(node[pathC[n-1]].y/normalizingFactor));
		glEnd();
	}
	
	glFlush();
}

void getInputFromUser(int* scope, int* numberOfNodes, int* numberOfRuns, int* maxNumberOfRows, int* maxNumberOfColumns, unsigned int* srandSeed, char *nodeFileName)
{
	*scope = -1;
	*numberOfNodes = -1;
	*numberOfRuns = -1;
	*maxNumberOfRows = -1;
	*maxNumberOfColumns = -1;
	
	printf("\n\n  What type run would you like to perform?");
	printf("\n  1 for one small randomly generated run.");
	printf("\n  2 for a series of small randomly generated runs.");
	printf("\n  3 for one on on a grid.");
	printf("\n  4 for a series of runs on randomly generated sized grids.");
	printf("\n  5 for one large randomly generated run.");
	printf("\n  6 for a series of large randomly generated runs.");
	printf("\n  7 to read nodes from nodeFile.");
	printf("\n  8 for a set created an self created function.");
	printf("\n\n  Inter an integer value: ");
	scanf("%d", scope);
	
	if(*scope == 1)
	{
		printf("\n\n  You will need to enter the number of nodes (Cities) wound you like to generate?");
		printf("\n  Note: If you choose a number bigger than 13 you may lock your computer up.");
		printf("\n        This is because we will be testing against the exaustive algorithim which has ((n-1)!)/2 paths.");
		
		printf("\n\n  You will also need to enter a seed to generate the random placed nodes");
		printf("\n  By intering the same seed on different exicutions of the program");
		printf("\n  you will be able to run multiple tests on the same node configuration.");
		
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of nodes) (seed)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", numberOfNodes, srandSeed);
		
		*numberOfRuns = 1;
	}
	else if(*scope == 2)
	{
		printf("\n\n  You will need to enter the number nodes (Cities) wound like to generate?");
		printf("\n  Note: If you choose a number bigger than 13 you may lock your computer up.");
		
		printf("\n\n  You will also need to enter how many randomly generated runs would you like to perform?");
		
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of nodes) (number of runs)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", numberOfNodes, numberOfRuns);
	}
	else if(*scope == 3)
	{
		printf("\n  You will need to enter the number of rows and columns for the grid you will generate.");
	
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of rows) (number of columns)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", maxNumberOfRows, maxNumberOfColumns);
		
		*numberOfRuns = 1;
	}
	else if(*scope == 4)
	{
		printf("\n  You will need to enter the number of rows and columns for the grid you will generate");
		printf("\n  and the number of runs");
	
		printf("\n  Enter three positive integer values seperated by a space: \n  (max number of rows) (max number of columns) (number of runs)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d %d", maxNumberOfRows, maxNumberOfColumns, numberOfRuns);
	}
	else if(*scope == 5)
	{
		printf("\n\n  You will need to enter the number of nodes (Cities) wound you like to generate?");
		
		printf("\n\n  You will also need to enter a seed to generate the random placed nodes");
		printf("\n  By intering the same seed on different exicutions of the program");
		printf("\n  you will be able to run multiple tests on the same node configuration.");
		
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of nodes) (seed)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", numberOfNodes, srandSeed);
		
		*numberOfRuns = 1;
	}
	else if(*scope == 6)
	{
		printf("\n\n  You will need to enter the number nodes (Cities) wound like to generate?");
		
		printf("\n\n  You will also need to enter how many randomly generated runs would you like to perform?");
		
		printf("\n  Enter two positive integer values seperated by a space: \n  (number of nodes) (number of runs)");
		printf("\n\n  Inter your values: ");
		scanf("%d %d", numberOfNodes, numberOfRuns);
	}
	else if(*scope == 7)
	{	
		*numberOfRuns = 1;
		
		printf("\n\n  You will need to enter the name of the file containing the nodes");
		
		printf("\n\n  The first element of the file should be an intiger containing the number of nodes.");
		printf("\n  The second element should be the value of the minumum path. This should be -1 if the value is unknown.");
		printf("\n  The following lines will contain the x y positions of all the nodes.");
		printf("\n\n  ");
		scanf("%s" , nodeFileName);
	
	}
	else if(*scope == 8)
	{	
		*numberOfRuns = 1;
	}
	else
	{
		printf("\n\n  Your input for type of run was invalide.");
		printf("\n\n  Good Bye.  \n\n");
		exit(0);
	}
}

void control()
{
	time_t t;
	int scope, numberOfNodes, numberOfRuns, maxNumberOfRows, maxNumberOfColumns;
	unsigned int srandSeed;
	int rows, columns, done;
	float4 *node;
	double nodeAdjustmentFactor;
	float4 geometricCenter;
	double distanceToOutermostNode;
	int *exhaustivePath, *nearestNeighborPath, *NbodyExtrusionPath;
	float4 *posNbody, *velNbody, *accNbody; 
	float *massNbody;
	double exhaustiveCost, nearestNeighborCost, NbodyExtrusionCost;
	int nodeCheck;
	double temp;
	char nodeFileName[100];
	
	getInputFromUser(&scope, &numberOfNodes, &numberOfRuns, &maxNumberOfRows, &maxNumberOfColumns, &srandSeed, nodeFileName);
	
	if(scope == 2 || scope == 4 || scope == 6 && PRINT_RAW_DATA_FILE == 1)
	{
		openRawDataFile(scope, numberOfRuns);
	}
	
	double totalNearestNeighborCost = 0.0;
	double totalNbodyExtrusionCost = 0.0;
	double totalPercentErrorNearestNeighbor = 0.0;
	double totalPercentErrorNbodyExtrusion = 0.0;
	double NbodyExtrusionVSNearestNeighbor = 0.0;
	
	for(int i = 0; i < numberOfRuns; i++)
	{	
		printf("\n\n\n  ********************* Intermediate Run %d ********************* ", i+1);
		
		exhaustiveCost = BIG_NUMBER;
		nearestNeighborCost = BIG_NUMBER;
		NbodyExtrusionCost = BIG_NUMBER;
		
		//Alocating memory
		if(scope == 3)
		{
			rows = maxNumberOfRows;
			columns = maxNumberOfColumns;
			numberOfNodes = rows*columns;
		}	
		if(scope == 4)
		{
			srand((unsigned) time(&t));
			done = -1;
			while(done == -1)
			{
				rows = 1 + (float)rand()/((float)RAND_MAX)*maxNumberOfRows;
				columns = 1 + (float)rand()/((float)RAND_MAX)*maxNumberOfColumns;
				printf("\n\n  rows = %d columns = %d", rows, columns);
				numberOfNodes = rows*columns;
				if(rows == 1 && columns == 1) done = -1;
				else done = 1;
			}
		}
		if(scope == 7)
		{
			getNumberOfNodesFromNodeFile(&numberOfNodes, &exhaustiveCost, nodeFileName);
		}
		if(scope == 8)
		{
			getNumberOfNodesFromASelfDefinedFunction(&numberOfNodes);
		}
		
		node = (float4*)malloc((numberOfNodes)*sizeof(float4));
	
		exhaustivePath = (int*)malloc((numberOfNodes)*sizeof(int));    // !!!!!!!!!!!! only needed in scope 1 and 2
		nearestNeighborPath = (int*)malloc((numberOfNodes)*sizeof(int));
		NbodyExtrusionPath = (int*)malloc((numberOfNodes)*sizeof(int));

		posNbody = (float4*)malloc((numberOfNodes)*sizeof(float4));
		velNbody = (float4*)malloc((numberOfNodes)*sizeof(float4));
		accNbody = (float4*)malloc((numberOfNodes)*sizeof(float4));
		massNbody = (float*)malloc((numberOfNodes)*sizeof(float4));
		
		//Creating nodes
		if(scope == 1 || scope == 2 || scope == 5 || scope == 6)
		{	
			placeNodesRandom(node, srandSeed, scope, numberOfNodes);
		}
		else if(scope == 3 || scope == 4)
		{
			placeNodesGrid(node, rows, columns);
		}
		else if(scope == 7)
		{
			placeNodesFromAFile(node, &numberOfNodes, nodeFileName);
		}
		else if(scope == 8)
		{
			placeNodesFromASelfDefinedFunction(node, numberOfNodes);
		}
		
		//Adjusting nodes
		geometricCenter = setGeometricCenterToZero(node, numberOfNodes);
		printf("\n\n  The geometric center of the nodes = (%f, %f)", geometricCenter.x, geometricCenter.y);
		
		distanceToOutermostNode = findDistanceToOuterMostElement(node, numberOfNodes);
		printf("\n  The distance to the outermost node from the geometric center pre adjustment is %f", distanceToOutermostNode);
		
		//nodeAdjustmentFactor = setAverageSeperationToOne(node, numberOfNodes);
		//nodeAdjustmentFactor = setMinimumSeperationToOne(node, numberOfNodes);
		nodeAdjustmentFactor = 1.0;
		printf("\n  The node adjustment factor = %f", nodeAdjustmentFactor);
		
		distanceToOutermostNode = findDistanceToOuterMostElement(node, numberOfNodes);
		printf("\n  The distance to the outermost node from the geometric center post adjustment is %f", distanceToOutermostNode);
		
		//Checking to see if a node is repeated
		nodeCheck = checkNodes(node, numberOfNodes);
		if(nodeCheck == -1)
		{
			printf("\n\n  There is a repeated node. Check your data set.");
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		
		//Drawing the adjusted nodes on the screen.
		drawPoints(node, numberOfNodes); 
		
		//Printing the edge costs (lengths in this case)
		if(PRINT_EDGE_COST == 1)
		{
			printEdgeCosts(node, numberOfNodes, nodeAdjustmentFactor);
		}
		
		//Finding exact cost
		printf("\n\n  Determining the exact cost.");
		if(scope == 1 || scope == 2)
		{	
			exhaustiveCost = exhaustiveTSP(node, exhaustivePath, numberOfNodes);
		}
		else if(scope == 3 || scope == 4)
		{
			//Assuming all edges are the same length. So just get the length of the first edge.
			temp = sqrt((node[0].x-node[1].x)*(node[0].x-node[1].x) + (node[0].y-node[1].y)*(node[0].y-node[1].y));
			if(rows == 1 || columns == 1)
			{
				exhaustiveCost = temp*2.0*(numberOfNodes - 1);
			}
			else if(rows%2 == 0 || columns%2 == 0)
			{
				exhaustiveCost = temp*numberOfNodes;
			}
			else
			{
				exhaustiveCost = temp*(numberOfNodes - 1.0 + sqrt(2.0));
			}
		}
		else if(scope == 5 || scope == 6)
		{
			exhaustiveCost = -1.0;
		}
		else if(scope == 7)
		{
			if(numberOfNodes < 13)
			{
				exhaustiveCost = exhaustiveTSP(node, exhaustivePath, numberOfNodes);
			}
		}
		else if(scope == 8)
		{
			if(numberOfNodes < 13)
			{
				exhaustiveCost = exhaustiveTSP(node, exhaustivePath, numberOfNodes);
			}
			else
			{
				exhaustiveCost = -1.0;
			}
		}
		printf("\n  Determining the exact cost is done.");
		
		//Finding nearest neighbor cost
		printf("\n\n  Running the nearest nieghbor algorithm.");
		nearestNeighborCost = nearestNeighborTSP(node, nearestNeighborPath, numberOfNodes);
		printf("\n  The nearest nieghbor algorithm is done.");
		
		//Running n-body extrusion code
		printf("\n\n  Running the N-body extrusion algorithm."); 
		printf("  \n"); //I had to enter this carage return so it would print the line above before it started the algorithm
		setNbodyInitailConditions(node, posNbody, velNbody, massNbody, numberOfNodes);
		NbodyExtrusionCost = NbodyExtrusionTSP(node, posNbody, velNbody, accNbody, massNbody, NbodyExtrusionPath, numberOfNodes);
		printf("  The N-body extrusion algorithm is done.");
		
		//Unadjusting costs
		exhaustiveCost *= nodeAdjustmentFactor;
		nearestNeighborCost *= nodeAdjustmentFactor;
		NbodyExtrusionCost *= nodeAdjustmentFactor;
		
		totalNearestNeighborCost += nearestNeighborCost;
		totalNbodyExtrusionCost += NbodyExtrusionCost;
		
		//Sanity check
		if(nearestNeighborCost < exhaustiveCost - SMALL_NUMBER)
		{
			printf("\n\n  Nearest Neighbor cost (%f) is smaller than exhaustive cost (%f). Something is wrong!\n",nearestNeighborCost, exhaustiveCost);
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		if(NbodyExtrusionCost < exhaustiveCost - SMALL_NUMBER)
		{
			printf("\n\n  Nbody Extrution cost (%f) is smaller than exhaustive cost (%f). Something is wrong!\n",NbodyExtrusionCost, exhaustiveCost);
			printf("\n\n  Good Bye.  \n\n");
			exit(0);
		}
		
		printf("\n\n  --------------------- Intermediate Run Results --------------------- ");
		
		// This is for debugging
		if(PRINT_PATHS == 1)
		{
			if(scope == 1 || scope == 2)
			{
				printf("\n\n  The exhaustive  path is     : "); 
				printPathOrder(exhaustivePath, numberOfNodes); 
				printf(" cost = %f", exhaustiveCost);
			}
			
			printf("\n\n  The nearest neighbor path is: "); 
			printPathOrder(nearestNeighborPath, numberOfNodes); 
			printf(" cost = %f", nearestNeighborCost);
			
			printf("\n\n  The Nbody extrusion path is : "); 
			printPathOrder(NbodyExtrusionPath, numberOfNodes); 
			printf(" cost = %f", NbodyExtrusionCost);
		}
		
		// Printing out the single run stats and acumulating the multiple run info to create final stats.
		// Stephen your stat collection should go here.
		if(exhaustiveCost < 0.0)
		{
			printf("\n\n  The minimum cost is unknown");
			printf("\n\n  The nearest neighbor cost is : %f", nearestNeighborCost);
			printf("\n\n  The Nbody extrusion cost is  : %f", NbodyExtrusionCost);
			NbodyExtrusionVSNearestNeighbor += (nearestNeighborCost - NbodyExtrusionCost)/nearestNeighborCost;
		}
		else
		{
			printf("\n\n  The minimum cost is          : %f ", exhaustiveCost);
			printf("\n\n  The nearest neighbor cost is : %f the precent error = %f", nearestNeighborCost, 100.0*(nearestNeighborCost - exhaustiveCost)/exhaustiveCost);
			printf("\n\n  The Nbody extrusion cost is  : %f the precent error = %f", NbodyExtrusionCost, 100.0*(NbodyExtrusionCost - exhaustiveCost)/exhaustiveCost);
			
			totalPercentErrorNearestNeighbor += 100.0*(nearestNeighborCost - exhaustiveCost)/exhaustiveCost;
			totalPercentErrorNbodyExtrusion  += 100.0*(NbodyExtrusionCost  - exhaustiveCost)/exhaustiveCost;
			
			NbodyExtrusionVSNearestNeighbor += (nearestNeighborCost - NbodyExtrusionCost)/nearestNeighborCost;
		}
		
		if(scope == 2 || scope == 6 && PRINT_RAW_DATA_FILE == 1)
		{
			fprintf(RawDataFile, "  %d, %d, %f, %f, %f\n", i+1, numberOfNodes, exhaustiveCost, nearestNeighborCost, NbodyExtrusionCost);
		}
		if(scope == 4 && PRINT_RAW_DATA_FILE == 1)
		{
			fprintf(RawDataFile, "  %d, %d, %d, %f, %f, %f\n", i+1, rows, columns, exhaustiveCost, nearestNeighborCost, NbodyExtrusionCost);
		}
	
		drawFInalPicture(node, exhaustivePath, nearestNeighborPath, NbodyExtrusionPath, scope, numberOfNodes);
		
		free(node);
		free(exhaustivePath);
		free(nearestNeighborPath);
		free(NbodyExtrusionPath);
		free(posNbody);
		free(velNbody);
		free(accNbody);
		free(massNbody);
	}
	
	printf("\n\n\n  $$$$$$$$$$$$$$$$$$$$$$$$$ Final results $$$$$$$$$$$$$$$$$$$$$$$$$$$$$");
	
	// Printing out the final acumulated stats.
	// Stephen your stat final stats should go here.
	if(exhaustiveCost < 0.0)
	{
		printf("\n\n  The average value of the nearest neighbor method was %f on %d run(s).", totalNearestNeighborCost/numberOfRuns, numberOfRuns);
		printf("\n\n  The average value of the Nbody extrution method was %f on %d run(s).", totalNbodyExtrusionCost/numberOfRuns, numberOfRuns);
	}
	else
	{
		printf("\n\n  The average percent error of the nearest neighbor method was %f on %d runs.", totalPercentErrorNearestNeighbor/(float)numberOfRuns, numberOfRuns);
		printf("\n  The average percent error of the Nbody extrution method was %f on %d runs", totalPercentErrorNbodyExtrusion/(float)numberOfRuns, numberOfRuns);
	}
	
	NbodyExtrusionVSNearestNeighbor = 100.0*NbodyExtrusionVSNearestNeighbor/(float)numberOfRuns;
	if(NbodyExtrusionVSNearestNeighbor >= 0)
	{
		printf("\n\n  The Nbody ectrusion method was on average %f percent better than the nearest neighbor method on %d run(s).", NbodyExtrusionVSNearestNeighbor, numberOfRuns);
	}
	else
	{
		printf("\n\n  The Nbody ectrusion method was on average %f percent worse than the nearest neighbor method on %d run(s).", -NbodyExtrusionVSNearestNeighbor, numberOfRuns);
	}
	
	if(scope == 2 || scope == 4 && PRINT_RAW_DATA_FILE == 1)
	{
		fclose(RawDataFile);
	}
	
	printf("\n\nDone\n");
	while(1);
}

int main(int argc, char** argv)
{
	glutInit(&argc,argv);
	glutInitWindowSize(X_WINDOW,Y_WINDOW);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Traveling Salesman Problem");
	glutDisplayFunc(control);
	glutMainLoop();
}



    

